#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <bitset>
#include <string>
#include <numeric>
#include <string>
#include <functional>
#include <cuda/pipeline>
#include <cuda/semaphore>

// need this to get tiled_partition > 32 threads
#define _CG_ABI_EXPERIMENTAL // enable experimental API

#include <hip/hip_cooperative_groups.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stq/gpu/queue.cuh>
#include <stq/gpu/aabb.cuh>
#include <stq/gpu/timer.cuh>
#include <stq/gpu/util.cuh>

using namespace std;
// using namespace cooperative_groups;
namespace cg = cooperative_groups;
typedef long long int ll;

__global__ void run(ll* in, ll* out, int N)
{
    __shared__ cuda::pipeline_shared_state<cuda::thread_scope_block, 2> pss;
    __shared__ Queue queue;
    queue.capacity = HEAP_SIZE;
    queue.heap_size = HEAP_SIZE;
    for (int i = threadIdx.x; i < HEAP_SIZE; i += blockDim.x) {
        queue.lock[i].release();
        queue.harr[i].x = -1.0; // release to add
                                // printf("Lock %i released\n", i);
    }
    __syncthreads();

    // Size must be a power of 2 and less than or equal to 32.
    cg::thread_block g = cg::this_thread_block();
    // thread_group tile32 = tiled_partition(g, 32);
    // int lane = tile32.thread_rank();
    // int tileIdx = g.thread_rank() / 32;

    // reserve shared memory for thread_block_tile usage.
    __shared__ cg::experimental::block_tile_memory<2, 1024> shared;
    cg::thread_block thb = cg::experimental::this_thread_block(shared);

    auto tilehalf = cg::experimental::tiled_partition<512>(thb);

    int lane = tilehalf.thread_rank();
    int lanerel = lane + blockIdx.x * blockDim.x;
    __syncthreads();

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    // if (tid >= N) return;
    // if in the first group + tid >= N
    if (tid >= N)
        return;

    // if (lane == 0)
    //     printf("group_rank: %i\n", tilehalf.meta_group_rank());

    if (tilehalf.meta_group_rank() == 0) {
        int curr1, curr2;
        int2 val1 = make_int2(in[tid], 0);
        int2 val2 = make_int2(0, in[tid]);
        curr1 = queue.push(lanerel, val1);
        curr2 = queue.push(lanerel, val2);
    } else {
        int2 res1 = queue.pop(lanerel % HEAP_SIZE);
        // int2 res2 = queue.pop(lanerel % HEAP_SIZE);
        out[tid] = res1.x;
    }

    //     int rand1, rand2;
    //     /* CUDA's random number library uses hiprandState_t to keep track of
    //     the seed value
    //      we will store a random state for every thread  */
    //     hiprandState_t state;

    //   /* we have to initialize the state */
    //     hiprand_init(0, /* the seed controls the sequence of random values
    //     that are produced */
    //                 0, /* the sequence number is only important with multiple
    //                 cores */ tid, /* the offset is how much extra we advance
    //                 in the sequence for each call, can be 0 */ &state);

    //   /* hiprand works like rand - except that it takes a state as a parameter
    //   */
    //     rand1 = int(hiprand(&state) % HEAP_SIZE);
    //     rand2 = int(hiprand(&state) % HEAP_SIZE);
    // if (tid == 0)
    //     printf("rand1: %i, rand2: %i\n", rand1, rand2);

    return;
}

int main(int argc, char** argv)
{
    vector<ll> nums;

    int N = atoi(argv[1]);

    for (ll i = 0; i < N; i++) {
        nums.push_back(i);
    }

    ll* d_in;
    hipMalloc((void**)&d_in, sizeof(ll) * N);
    hipMemcpy(d_in, nums.data(), sizeof(ll) * N, hipMemcpyHostToDevice);

    ll* d_out;
    hipMalloc((void**)&d_out, sizeof(ll) * N);
    hipMemset(d_out, 0, sizeof(ll) * N);

    int block = 1024;
    int grid = (N / block + 1);
    printf("grid size: %i\n", grid);
    printf(
        "sizeof(semaphore):  %i\n",
        sizeof(cuda::binary_semaphore<cuda::thread_scope_block>));
    printf("sizeof(int2):  %i\n", sizeof(int2));

    recordLaunch("run", grid, block, 8, run, d_in, d_out, N);
    hipDeviceSynchronize();

    vector<ll> out;
    out.resize(N);
    hipMemcpy(out.data(), d_out, sizeof(ll) * N, hipMemcpyDeviceToHost);

    // int s = accumulate(out.begin(), out.end(), 0);
    int s = 0;
    for (int i = 0; i < N; i += 1) {
        // printf("%lld:%lld ", nums[i], out[i]);
        s += out[i];
    }
    printf("\n");
    printf("sum: %i\n", s);
}