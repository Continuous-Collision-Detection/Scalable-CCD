#include "hip/hip_runtime.h"
#include "root_finder.cuh"

#include <scalable_ccd/config.hpp>
#include <scalable_ccd/cuda/narrow_phase/ccd_buffer.cuh>
#include <scalable_ccd/cuda/narrow_phase/ccd_config.cuh>
#include <scalable_ccd/cuda/narrow_phase/ccd_data.cuh>
#include <scalable_ccd/cuda/narrow_phase/interval.cuh>
#include <scalable_ccd/cuda/utils/assert.cuh>
#include <scalable_ccd/cuda/utils/atomic_min_float.cuh>
#include <scalable_ccd/cuda/utils/device_variable.cuh>
#include <scalable_ccd/utils/logger.hpp>

#include <vector>

namespace scalable_ccd::cuda {

namespace {
    // Allocate this in constant memory for faster access
    __device__ __constant__ CCDConfig CONFIG;

    __device__ inline bool
    sum_less_than_one(const Scalar& num1, const Scalar& num2)
    {
#ifdef SCALABLE_CCD_USE_DOUBLE
        return num1 + num2 <= 1 / (1 - DBL_EPSILON);
#else
        return num1 + num2 <= 1 / (1 - FLT_EPSILON);
#endif
    }

    __device__ inline Scalar max_Linf_4(
        const Vector3& p1,
        const Vector3& p2,
        const Vector3& p3,
        const Vector3& p4,
        const Vector3& p1e,
        const Vector3& p2e,
        const Vector3& p3e,
        const Vector3& p4e)
    {
        return max(
            max((p1e - p1).lpNorm<Eigen::Infinity>(),
                (p2e - p2).lpNorm<Eigen::Infinity>()),
            max((p3e - p3).lpNorm<Eigen::Infinity>(),
                (p4e - p4).lpNorm<Eigen::Infinity>()));
    }

    __device__ void compute_face_vertex_tolerance(CCDData& data_in)
    {
        const Vector3 p000 = data_in.v0s - data_in.v1s;
        const Vector3 p001 = data_in.v0s - data_in.v3s;
        const Vector3 p011 =
            data_in.v0s - (data_in.v2s + data_in.v3s - data_in.v1s);
        const Vector3 p010 = data_in.v0s - data_in.v2s;
        const Vector3 p100 = data_in.v0e - data_in.v1e;
        const Vector3 p101 = data_in.v0e - data_in.v3e;
        const Vector3 p111 =
            data_in.v0e - (data_in.v2e + data_in.v3e - data_in.v1e);
        const Vector3 p110 = data_in.v0e - data_in.v2e;

        data_in.tol[0] = CONFIG.co_domain_tolerance
            / (3 * max_Linf_4(p000, p001, p011, p010, p100, p101, p111, p110));
        data_in.tol[1] = CONFIG.co_domain_tolerance
            / (3 * max_Linf_4(p000, p100, p101, p001, p010, p110, p111, p011));
        data_in.tol[2] = CONFIG.co_domain_tolerance
            / (3 * max_Linf_4(p000, p100, p110, p010, p001, p101, p111, p011));
    }

    __device__ void compute_edge_edge_tolerance(CCDData& data_in)
    {
        // WARNING: This differs from the original implementation in
        // Tight-Inclusion! This may be a bug, but results in better accuracy.
        const Vector3 p000 = data_in.v0s - data_in.v2s;
        const Vector3 p001 = data_in.v0s - data_in.v3s;
        const Vector3 p010 = data_in.v1s - data_in.v2s;
        const Vector3 p011 = data_in.v1s - data_in.v3s;
        const Vector3 p100 = data_in.v0e - data_in.v2e;
        const Vector3 p101 = data_in.v0e - data_in.v3e;
        const Vector3 p110 = data_in.v1e - data_in.v2e;
        const Vector3 p111 = data_in.v1e - data_in.v3e;

        data_in.tol[0] = CONFIG.co_domain_tolerance
            / (3 * max_Linf_4(p000, p001, p011, p010, p100, p101, p111, p110));
        data_in.tol[1] = CONFIG.co_domain_tolerance
            / (3 * max_Linf_4(p000, p001, p011, p010, p100, p101, p111, p110));
        data_in.tol[2] = CONFIG.co_domain_tolerance
            / (3 * max_Linf_4(p000, p100, p101, p001, p010, p110, p111, p011));
    }

    template <bool is_vf>
    __device__ __host__ void get_numerical_error(CCDData& data_in, bool use_ms)
    {
        Scalar filter;
        if (!use_ms) {
#ifdef SCALABLE_CCD_USE_DOUBLE
            if constexpr (is_vf) {
                filter = 6.661338147750939e-15;
            } else {
                filter = 6.217248937900877e-15;
            }
#else
            if constexpr (is_vf) {
                filter = 3.576279e-06;
            } else {
                filter = 3.337861e-06;
            }
#endif
        } else {
#ifdef SCALABLE_CCD_USE_DOUBLE
            if constexpr (is_vf) {
                filter = 7.549516567451064e-15;
            } else {
                filter = 7.105427357601002e-15;
            }
#else
            if constexpr (is_vf) {
                filter = 4.053116e-06;
            } else {
                filter = 3.814698e-06;
            }
#endif
        }

        const Vector3 max = data_in.v0s.cwiseAbs()
                                .cwiseMax(data_in.v1s.cwiseAbs())
                                .cwiseMax(data_in.v2s.cwiseAbs())
                                .cwiseMax(data_in.v3s.cwiseAbs())
                                .cwiseMax(data_in.v0e.cwiseAbs())
                                .cwiseMax(data_in.v1e.cwiseAbs())
                                .cwiseMax(data_in.v2e.cwiseAbs())
                                .cwiseMax(data_in.v3e.cwiseAbs())
                                .cwiseMax(Vector3::Ones());

        data_in.err = max.array() * max.array() * max.array() * filter;
    }

    __device__ Vector3
    calculate_vf(const CCDData& data_in, const DomainCorner& tuv)
    {
        const Vector3 v = (data_in.v0e - data_in.v0s) * tuv.t + data_in.v0s;
        const Vector3 t0 = (data_in.v1e - data_in.v1s) * tuv.t + data_in.v1s;
        const Vector3 t1 = (data_in.v2e - data_in.v2s) * tuv.t + data_in.v2s;
        const Vector3 t2 = (data_in.v3e - data_in.v3s) * tuv.t + data_in.v3s;
        return v - (t1 - t0) * tuv.u - (t2 - t0) * tuv.v - t0;
    }

    __device__ Vector3
    calculate_ee(const CCDData& data_in, const DomainCorner& tuv)
    {
        const Vector3 ea0 = (data_in.v0e - data_in.v0s) * tuv.t + data_in.v0s;
        const Vector3 ea1 = (data_in.v1e - data_in.v1s) * tuv.t + data_in.v1s;
        const Vector3 eb0 = (data_in.v2e - data_in.v2s) * tuv.t + data_in.v2s;
        const Vector3 eb1 = (data_in.v3e - data_in.v3s) * tuv.t + data_in.v3s;
        return ((ea1 - ea0) * tuv.u + ea0) - ((eb1 - eb0) * tuv.v + eb0);
    }

    template <bool is_vf>
    __device__ bool origin_in_inclusion_function(
        const CCDData& data_in,
        const CCDDomain& domain,
        Scalar& true_tol,
        bool& box_in)
    {
        Array3 codomain_min = Array3::Constant(SCALAR_MAX);
        Array3 codomain_max = Array3::Constant(-SCALAR_MAX);

        DomainCorner domain_corner;
        for (uint8_t corner = 0; corner < 8; corner++) {
            domain_corner.update_tuv(domain, corner);

            Vector3 codomain_corner;
            if constexpr (is_vf) {
                codomain_corner = calculate_vf(data_in, domain_corner);
            } else {
                codomain_corner = calculate_ee(data_in, domain_corner);
            }

            codomain_min = codomain_min.min(codomain_corner.array());
            codomain_max = codomain_max.max(codomain_corner.array());
        }

        // get the min and max in one dimension
        true_tol = max(0.0, (codomain_max - codomain_min).maxCoeff());

        box_in = true;

        if ((codomain_min - data_in.ms > data_in.err).any()
            || (codomain_max + data_in.ms < -data_in.err).any()) {
            return false;
        }

        if ((codomain_min + data_in.ms < -data_in.err).any()
            || (codomain_max - data_in.ms > data_in.err).any()) {
            box_in = false;
        }

        return true;
    }

    __device__ int split_dimension(const CCDData& data, Array3 width)
    {
        const Array3 res = width / data.tol;
        if (res[0] >= res[1] && res[0] >= res[2]) {
            return 0;
        } else if (res[1] >= res[0] && res[1] >= res[2]) {
            return 1;
        } else {
            assert(res[2] >= res[0] && res[2] >= res[1]);
            return 2;
        }
    }

    template <bool is_vf>
    __device__ inline bool bisect(
        const CCDDomain& domain,
        const int split,
        const Scalar* const toi,
        CCDBuffer* const buffer)
    {
        const SplitInterval halves(domain.tuv[split]);

        if (halves.first.lower >= halves.first.upper
            || halves.second.lower >= halves.second.upper) {
            return true;
        }

        buffer->push(domain).tuv[split] = halves.first;

        if (split == 0) {
            if (halves.second.lower <= *toi) {
                buffer->push(domain).tuv[0] = halves.second;
            }
        } else {
            if constexpr (is_vf) {
                if (split == 1) {
                    // check if u+v<=1
                    if (sum_less_than_one(
                            halves.second.lower, domain.tuv[2].lower)) {
                        buffer->push(domain).tuv[1] = halves.second;
                    }
                } else if (split == 2) {
                    // check if u+v<=1
                    if (sum_less_than_one(
                            halves.second.lower, domain.tuv[1].lower)) {
                        buffer->push(domain).tuv[2] = halves.second;
                    }
                }
            } else {
                buffer->push(domain).tuv[split] = halves.second;
            }
        }

        return false;
    }

} // namespace

// === the memory pool method =================================================

template <bool is_vf>
__global__ void compute_tolerance(CCDData* data, const int query_size)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx >= query_size)
        return;

    if constexpr (is_vf) {
        compute_face_vertex_tolerance(data[tx]);
    } else {
        compute_edge_edge_tolerance(data[tx]);
    }

    data[tx].nbr_checks = 0;
    get_numerical_error<is_vf>(data[tx], CONFIG.use_ms);
}

template <bool is_vf>
__global__ void
ccd_kernel(CCDBuffer* const buffer, CCDData* const data, Scalar* const toi)
{
    const int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx >= buffer->starting_size())
        return;

    // Get the tx element from the buffer without advancing the head.
    // This allows for better coalescing of memory access compared to pop().
    const CCDDomain domain_in = (*buffer)[tx];
    const CCDData data_in = data[domain_in.query_id];
    atomicAdd(&data[domain_in.query_id].nbr_checks, 1);

    const Scalar min_t = domain_in.tuv[0].lower; // the time of this domain

#ifndef SCALABLE_CCD_TOI_PER_QUERY
    // if the time is larger than toi, return
    if (min_t >= *toi) {
#else
    if (min_t >= data_in.toi) {
#endif
        return;
    }

    // Check if exceeds max checks
    if (CONFIG.max_iter >= 0 && data_in.nbr_checks > CONFIG.max_iter) {
        return;
    }
    // else if (buffer.mp_remaining > buffer->capacity() / 2) { // overflow
    //     atomicCAS(&buffer->overflow_flag, 0, 1);
    //     return;
    // }

    Scalar true_tol = 0; // set by origin_in_inclusion_function
    bool box_in;         // set by origin_in_inclusion_function
    if (origin_in_inclusion_function<is_vf>(
            data_in, domain_in, true_tol, box_in)) {

        const Array3 widths(
            domain_in.tuv[0].upper - domain_in.tuv[0].lower,
            domain_in.tuv[1].upper - domain_in.tuv[1].lower,
            domain_in.tuv[2].upper - domain_in.tuv[2].lower);

        // Condition 1: the domain is smaller than the tolerance.
        if ((widths <= data_in.tol).all()) {
            atomicMin(toi, min_t);
#ifdef SCALABLE_CCD_TOI_PER_QUERY
            atomicMin(&data[domain_in.query_id].toi, min_t);
#endif
            return;
        }

        // Condition 2: the box is inside the epsilon box
        if (box_in && (CONFIG.allow_zero_toi || min_t > 0)) {
            atomicMin(toi, min_t);
#ifdef SCALABLE_CCD_TOI_PER_QUERY
            atomicMin(&data[domain_in.query_id].toi, min_t);
#endif
            return;
        }

        // Condition 3: real tolerance is smaller than the int tolerance
        if (true_tol <= CONFIG.co_domain_tolerance
            && (CONFIG.allow_zero_toi || min_t > 0)) {
            atomicMin(toi, min_t);
#ifdef SCALABLE_CCD_TOI_PER_QUERY
            atomicMin(&data[domain_in.query_id].toi, min_t);
#endif
            return;
        }

        // Get the next dimension to split
        const int split = split_dimension(data_in, widths);

        const bool sure_in = bisect<is_vf>(
            domain_in, split,
#ifdef SCALABLE_CCD_TOI_PER_QUERY
            &data_in.toi,
#else
            toi,
#endif
            buffer);

        // Condition 4 (rare): the interval is too small that overflow happens
        if (sure_in) {
            atomicMin(toi, min_t);
#ifdef SCALABLE_CCD_TOI_PER_QUERY
            atomicMin(&data[domain_in.query_id].toi, min_t);
#endif
            return;
        }
    }
}

template <bool is_vf>
bool ccd(
    thrust::device_vector<CCDData>& d_data,
    const std::shared_ptr<MemoryHandler> memory_handler,
    const int parallel_nbr,
    const int max_iter,
    const Scalar tol,
    const bool use_ms,
    const bool allow_zero_toi,
    Scalar& toi)
{
    const int nbr = d_data.size();

    CCDBuffer* d_buffer;
    {
        const size_t unit_size = memory_handler->MAX_UNIT_SIZE;

        logger().trace(
            "CCD Buffer of size {:d} ({:g} GB)", unit_size,
            sizeof(CCDDomain) * unit_size / 1e9);

        CCDBuffer buffer;
        gpuErrchk(hipMalloc(&buffer.m_data, sizeof(CCDDomain) * unit_size));
        buffer.m_starting_size = nbr;
        buffer.m_capacity = unit_size;
        buffer.m_head = 0;
        buffer.m_tail = nbr;
        buffer.m_overflow_flag = 0;

        gpuErrchk(hipMalloc(&d_buffer, sizeof(CCDBuffer)));
        gpuErrchk(hipMemcpy(
            d_buffer, &buffer, sizeof(CCDBuffer), hipMemcpyHostToDevice));

        initialize_buffer<<<nbr / parallel_nbr + 1, parallel_nbr>>>(d_buffer);
        gpuErrchk(hipDeviceSynchronize());
    }

    // Initialize the global configuration variable
    {
        CCDConfig config;
        config.co_domain_tolerance = tol;
        config.use_ms = use_ms;
        config.allow_zero_toi = allow_zero_toi;
        config.max_iter = max_iter;
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(CONFIG), &config, sizeof(CCDConfig)));
    }

    DeviceVariable d_toi(toi);

    // ---

    compute_tolerance<is_vf><<<nbr / parallel_nbr + 1, parallel_nbr>>>(
        thrust::raw_pointer_cast(d_data.data()), nbr);
    gpuErrchk(hipDeviceSynchronize());

    logger().trace("Max queries: {:d}", memory_handler->MAX_QUERIES);

    int nbr_per_loop = nbr;
    logger().trace("Queue size t0: {:d}", nbr_per_loop);
    while (nbr_per_loop > 0) {
        ccd_kernel<is_vf><<<nbr_per_loop / parallel_nbr + 1, parallel_nbr>>>(
            d_buffer, thrust::raw_pointer_cast(d_data.data()), &d_toi);
        gpuErrchk(hipDeviceSynchronize());

        // Update the starting index to reflect that the starting_size was
        // procesed.
        shift_queue_start<<<1, 1>>>(d_buffer);
        gpuErrchk(hipDeviceSynchronize());

        gpuErrchk(hipMemcpy(
            &nbr_per_loop, &(d_buffer->m_starting_size), sizeof(unsigned),
            hipMemcpyDeviceToHost));

        logger().trace("Queue size: {:d}", nbr_per_loop);
        logger().trace("toi={:g}", *d_toi);
    }
    gpuErrchk(hipDeviceSynchronize());

    toi = d_toi;

    int overflow;
    gpuErrchk(hipMemcpy(
        &overflow, &(d_buffer->m_overflow_flag), sizeof(int),
        hipMemcpyDeviceToHost));
    return overflow;
}

// === Template instantiation ==================================================

template __global__ void compute_tolerance<false>(CCDData*, const int);
template __global__ void compute_tolerance<true>(CCDData*, const int);

template __global__ void
ccd_kernel<false>(CCDBuffer* const, CCDData* const, Scalar* const);
template __global__ void
ccd_kernel<true>(CCDBuffer* const, CCDData* const, Scalar* const);

// clang-format off
template bool ccd<false>(
    thrust::device_vector<CCDData>&, const std::shared_ptr<MemoryHandler>,
    const int, const int, const Scalar, const bool, const bool, Scalar&);
template bool ccd<true>(
    thrust::device_vector<CCDData>&, const std::shared_ptr<MemoryHandler>,
    const int, const int, const Scalar, const bool, const bool, Scalar&);
// clang-format on

} // namespace scalable_ccd::cuda
