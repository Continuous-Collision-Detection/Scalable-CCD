#include "hip/hip_runtime.h"
#include "narrow_phase.cuh"

#include <scalable_ccd/config.hpp>

#include <fstream>
#include <iostream>

#include <scalable_ccd/cuda/narrow_phase/root_finder.cuh>
#include <scalable_ccd/cuda/broad_phase/aabb.cuh>
#include <scalable_ccd/cuda/utils/assert.cuh>
#include <scalable_ccd/cuda/utils/device_variable.cuh>
#include <scalable_ccd/cuda/utils/device_buffer.cuh>
#include <scalable_ccd/utils/profiler.hpp>
#include <scalable_ccd/utils/logger.hpp>

namespace scalable_ccd::cuda {

namespace {

    /// @brief Split the heterogeneous array of overlaps into two array of vertex-face and edge-edge overlaps.
    /// @param[in] boxes The array of AABBs
    /// @param[in] overlaps The array of pairs of indices of the boxes that overlap
    /// @param[in] n_overlaps The number of overlaps
    /// @param[out] vf_overlaps The output array of pairs of indices of the vertex-face overlaps
    /// @param[out] ee_overlaps The output array of pairs of indices of the edge-edge overlaps
    __global__ void split_overlaps(
        const AABB* const boxes,
        const int2* const overlaps,
        const int n_overlaps,
        RawDeviceBuffer<int2> vf_overlaps,
        RawDeviceBuffer<int2> ee_overlaps)
    {
        const int tid = threadIdx.x + blockIdx.x * blockDim.x;
        if (tid >= n_overlaps)
            return;

        const int a_id = min(overlaps[tid].x, overlaps[tid].y);
        const int b_id = max(overlaps[tid].x, overlaps[tid].y);

        if (boxes[a_id].is_vertex() && boxes[b_id].is_face()) {
            vf_overlaps.push(make_int2(a_id, b_id));
        } else if (boxes[a_id].is_edge() && boxes[b_id].is_edge()) {
            ee_overlaps.push(make_int2(a_id, b_id));
        } else {
            printf(
                "Invalid overlap: %d %d [(%d %d %d); (%d %d %d)]\n", a_id, b_id,
                boxes[a_id].is_vertex(), boxes[a_id].is_edge(),
                boxes[a_id].is_face(), boxes[b_id].is_vertex(),
                boxes[b_id].is_edge(), boxes[b_id].is_face());
            assert(false);
        }
    }

    /// @brief Populate the CCDData array with the necessary data for the narrow phase.
    /// @param V0 Vertex positions at time t=0
    /// @param V1 Vertex positions at time t=1
    /// @param n_vertices The number of vertices
    /// @param boxes The array of AABBs
    /// @param overlaps The array of pairs of indices of the boxes that overlap
    /// @param ms Minimum separation distance
    /// @param data The output array of CCDData
    __global__ void add_data(
        const Scalar* const V0,
        const Scalar* const V1,
        const int n_vertices,
        const AABB* const boxes,
        const RawDeviceBuffer<int2> overlaps,
        const Scalar ms,
        CCDData* data)
    {
        const int tid = threadIdx.x + blockIdx.x * blockDim.x;
        if (tid >= *overlaps.size)
            return;

        data[tid].ms = ms;

        const int minner = min(overlaps[tid].x, overlaps[tid].y);
        const int maxxer = max(overlaps[tid].x, overlaps[tid].y);
        const int3 avids = boxes[minner].vertex_ids;
        const int3 bvids = boxes[maxxer].vertex_ids;

#ifdef SCALABLE_CCD_TOI_PER_QUERY
        data[tid].toi = std::numeric_limits<Scalar>::infinity();
        // data[tid].box_id = shift + tid;
        data[tid].aid = minner;
        data[tid].bid = maxxer;
#endif

        if (AABB::is_vertex(avids) && AABB::is_face(bvids)) {
            for (size_t i = 0; i < 3; i++) {
                data[tid].v0s[i] = V0[avids.x + i * n_vertices];
                data[tid].v1s[i] = V0[bvids.x + i * n_vertices];
                data[tid].v2s[i] = V0[bvids.y + i * n_vertices];
                data[tid].v3s[i] = V0[bvids.z + i * n_vertices];
                data[tid].v0e[i] = V1[avids.x + i * n_vertices];
                data[tid].v1e[i] = V1[bvids.x + i * n_vertices];
                data[tid].v2e[i] = V1[bvids.y + i * n_vertices];
                data[tid].v3e[i] = V1[bvids.z + i * n_vertices];
            }
        } else if (AABB::is_edge(avids) && AABB::is_edge(bvids)) {
            for (size_t i = 0; i < 3; i++) {
                data[tid].v0s[i] = V0[avids.x + i * n_vertices];
                data[tid].v1s[i] = V0[avids.y + i * n_vertices];
                data[tid].v2s[i] = V0[bvids.x + i * n_vertices];
                data[tid].v3s[i] = V0[bvids.y + i * n_vertices];
                data[tid].v0e[i] = V1[avids.x + i * n_vertices];
                data[tid].v1e[i] = V1[avids.y + i * n_vertices];
                data[tid].v2e[i] = V1[bvids.x + i * n_vertices];
                data[tid].v3e[i] = V1[bvids.y + i * n_vertices];
            }
        } else {
            assert(false);
        }
    }

} // namespace

void run_narrow_phase(
    const DeviceMatrix<Scalar>& d_vertices_t0,
    const DeviceMatrix<Scalar>& d_vertices_t1,
    const thrust::device_vector<AABB>& d_boxes,
    const thrust::device_vector<int2>& d_overlaps,
    const int threads,
    const int max_iter,
    const Scalar tol,
    const Scalar ms,
    const bool allow_zero_toi,
    std::shared_ptr<MemoryHandler> memory_handler,
    std::vector<int>& result_list,
    Scalar& toi)
{
    assert(toi >= 0);

    const bool use_ms = ms > 0;

    size_t start_id = 0;
    size_t size = d_overlaps.size();
    memory_handler->MAX_QUERIES = size;

    size_t remaining_queries;
#ifndef SCALABLE_CCD_TOI_PER_QUERY
    while ((remaining_queries = size - start_id) > 0 && toi > 0) {
#else
    while ((remaining_queries = size - start_id) > 0) {
#endif
        logger().trace("Remaining queries to process: {:d}", remaining_queries);

        bool overflowed = false;
        size_t n_queries_to_process;

        do {
            n_queries_to_process =
                std::min(remaining_queries, memory_handler->MAX_QUERIES);

            if (!overflowed) { // only true in first iteration
                memory_handler->handleNarrowPhase(n_queries_to_process);
            } else {
                memory_handler->handleOverflow(n_queries_to_process);
            }
            assert(n_queries_to_process > 0);
            assert(n_queries_to_process <= d_overlaps.size());

            thrust::device_vector<CCDData> d_vf_data_list, d_ee_data_list;
            {
                // Allocate enough space for the worst case
                DeviceBuffer<int2> d_vf_overlaps(n_queries_to_process);
                DeviceBuffer<int2> d_ee_overlaps(n_queries_to_process);

                {
                    SCALABLE_CCD_GPU_PROFILE_POINT("splitOverlaps");

                    split_overlaps<<<
                        n_queries_to_process / threads + 1, threads>>>(
                        thrust::raw_pointer_cast(d_boxes.data()),
                        thrust::raw_pointer_cast(d_overlaps.data()) + start_id,
                        n_queries_to_process, d_vf_overlaps, d_ee_overlaps);

                    gpuErrchk(hipDeviceSynchronize());
                }

                logger().trace(
                    "# FV queries: {:d}; # EE queries: {:d}",
                    d_vf_overlaps.size(), d_ee_overlaps.size());

                {
                    SCALABLE_CCD_GPU_PROFILE_POINT("createDataList");

                    d_vf_data_list.resize(d_vf_overlaps.size());
                    add_data<<<d_vf_data_list.size() / threads + 1, threads>>>(
                        d_vertices_t0.data(), d_vertices_t1.data(),
                        d_vertices_t0.rows(),
                        thrust::raw_pointer_cast(d_boxes.data()), d_vf_overlaps,
                        ms, thrust::raw_pointer_cast(d_vf_data_list.data()));
                    gpuErrchk(hipDeviceSynchronize());

                    d_ee_data_list.resize(d_ee_overlaps.size());
                    add_data<<<d_ee_data_list.size() / threads + 1, threads>>>(
                        d_vertices_t0.data(), d_vertices_t1.data(),
                        d_vertices_t0.rows(),
                        thrust::raw_pointer_cast(d_boxes.data()), d_ee_overlaps,
                        ms, thrust::raw_pointer_cast(d_ee_data_list.data()));
                    gpuErrchk(hipDeviceSynchronize());
                }
            }

            logger().trace(
                "Narrow phase CCD data size: {:g} GB",
                (d_vf_data_list.size() + d_ee_data_list.size())
                    * sizeof(CCDData) / 1e9);

            constexpr int parallel = 64;
            logger().trace(
                "Running memory-pooled CCD using {:d} threads", parallel);
            {
                SCALABLE_CCD_GPU_PROFILE_POINT(
                    "run_memory_pool_ccd (narrowphase)");

                overflowed = run_memory_pool_ccd(
                    d_vf_data_list, memory_handler, /*is_edge_edge=*/false,
                    result_list, parallel, max_iter, tol, use_ms,
                    allow_zero_toi, toi);

                gpuErrchk(hipDeviceSynchronize());
            }

            if (overflowed) // rerun
            {
                logger().debug(
                    "Narrow-phase: overflowed upon face-vertex; reducing parallel count");
                continue;
            }

            logger().debug("ToI after FV: {:e}", toi);

            {
                SCALABLE_CCD_GPU_PROFILE_POINT(
                    "run_memory_pool_ccd (narrowphase)");

                overflowed = run_memory_pool_ccd(
                    d_ee_data_list, memory_handler, /*is_edge_edge=*/true,
                    result_list, parallel, max_iter, tol, use_ms,
                    allow_zero_toi, toi);

                gpuErrchk(hipDeviceSynchronize());
            }

            if (overflowed) {
                logger().debug(
                    "Narrow-phase: overflowed upon edge-edge; reducing parallel count");
            }

            logger().debug("ToI after EE: {:e}", toi);
        } while (overflowed);

        start_id += n_queries_to_process;
    }
}

#if false
void run_ccd(
    const std::vector<AABB>& boxes,
    std::shared_ptr<MemoryHandler> memory_handler,
    const Eigen::MatrixXd& _vertices_t0,
    const Eigen::MatrixXd& _vertices_t1,
    int N,
    int& nbox,
    int& parallel,
    int& devcount,
    int& limitGB,
    std::vector<std::pair<int, int>>& overlaps,
    std::vector<int>& result_list,
    const bool allow_zero_toi,
    Scalar& min_distance,
    Scalar& toi)
{
    assert(_vertices_t0.rows() == _vertices_t1.rows());
    assert(_vertices_t0.cols() == _vertices_t1.cols());

    constexpr int bpthreads = 32; // TODO: hardcoding threads for now
    constexpr int npthreads = 1024;

    // Const variables
#ifdef SCALABLE_CCD_USE_DOUBLE
    const Eigen::MatrixXd& vertices_t0 = _vertices_t0;
    const Eigen::MatrixXd& vertices_t1 = _vertices_t1;
#else
    const Eigen::MatrixXf vertices_t0 = _vertices_t0.cast<float>();
    const Eigen::MatrixXf vertices_t1 = _vertices_t1.cast<float>();
#endif
    logger().trace("Copying vertices");
    thrust::device_vector<Scalar> d_vertices_t0, d_vertices_t1;
    thrust::copy(
        vertices_t0.data(), vertices_t0.data() + vertices_t0.size(),
        d_vertices_t0.begin());
    thrust::copy(
        vertices_t1.data(), vertices_t1.data() + vertices_t1.size(),
        d_vertices_t1.begin());

    toi = 1;

    int tidstart = 0;

    size_t total_count = 0;
    while (N > tidstart && toi > 0) {
        logger().trace(
            "Next loop: N {:d}, tidstart {:d}", boxes.size(), tidstart);

        r.Start("runBroadPhase", /*gpu=*/true);
        thrust::device_vector<int2> d_overlaps;
        runBroadPhase(
            boxes, memory_handler, nbox, overlaps, d_overlaps, bpthreads,
            tidstart, devcount, limitGB);
        r.Stop();

        logger().trace("First run end {:d}", tidstart);
        // memory_handler->increaseOverlapCutoff(2);
        logger().trace("Next cutoff {:d}", memory_handler->MAX_OVERLAP_CUTOFF);

        logger().trace("Threads now {:d}", npthreads);

        r.Start("copyBoxesToGpu", /*gpu=*/true);

        total_count += d_overlaps.size();
        logger().trace("Count {:d}", d_overlaps.size());

        AABB* d_boxes = copy_to_gpu(boxes.data(), boxes.size());
        r.Stop();

        r.Start("copyVerticesToGpu", /*gpu=*/true);
        logger().trace("Copying vertices");
        double* d_vertices_t0 =
            copy_to_gpu(vertices_t0.data(), vertices_t0.size());
        double* d_vertices_t1 =
            copy_to_gpu(vertices_t1.data(), vertices_t1.size());
        r.Stop();
        int n_vertices = vertices_t0.rows();
        assert(n_vertices == vertices_t1.rows());

        int max_iter = -1;
        Scalar tolerance = 1e-6;

        run_narrowphase(
            d_overlaps, d_boxes, memory_handler, count, d_vertices_t0,
            d_vertices_t1, n_vertices, npthreads, max_iter, tolerance, ms,
            allow_zero_toi, result_list, toi, r);
        gpuErrchk(hipGetLastError());

        gpuErrchk(hipFree(d_count));
        gpuErrchk(hipFree(d_overlaps));
        gpuErrchk(hipFree(d_boxes));
        gpuErrchk(hipFree(d_vertices_t0));
        gpuErrchk(hipFree(d_vertices_t1));

        gpuErrchk(hipGetLastError());

        hipDeviceSynchronize();
    }
    logger().info("Total count {:d}", tot_count);
    logger().info("LimitGB {:d}", memory_handler->limitGB);
}
#endif

} // namespace scalable_ccd::cuda