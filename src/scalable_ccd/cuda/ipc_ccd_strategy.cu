#include "ipc_ccd_strategy.hpp"

#include <scalable_ccd/config.hpp>
#include <scalable_ccd/cuda/broad_phase/broad_phase.cuh>
#include <scalable_ccd/cuda/narrow_phase/narrow_phase.cuh>
#include <scalable_ccd/cuda/utils/device_matrix.cuh>
#include <scalable_ccd/cuda/utils/assert.cuh>
#include <scalable_ccd/utils/profiler.hpp>

namespace scalable_ccd::cuda {
namespace {
    template <bool run_vf>
    void partial_ipc_ccd_strategy(
        const DeviceMatrix<Scalar>& d_vertices_t0,
        const DeviceMatrix<Scalar>& d_vertices_t1,
        const DeviceMatrix<int>& d_edges,
        const DeviceMatrix<int>& d_faces,
        const std::shared_ptr<DeviceAABBs> d_vertex_boxes,
        const std::shared_ptr<DeviceAABBs> d_edge_boxes,
        const std::shared_ptr<DeviceAABBs> d_face_boxes,
        const Scalar min_distance,
        const int max_iterations,
        const Scalar tolerance,
        Scalar& earliest_toi)
    {
        constexpr int np_threads = 1024;

        // --- Build broad phase ----------------------------------------------
        logger().trace("Building broad phase");

        std::shared_ptr<MemoryHandler> memory_handler =
            std::make_shared<MemoryHandler>();

        BroadPhase broad_phase(memory_handler);
        if constexpr (run_vf) {
            broad_phase.build(d_vertex_boxes, d_face_boxes);
        } else {
            broad_phase.build(d_edge_boxes);
        }

        // --- Run broad + narrow phase ---------------------------------------

        while (!broad_phase.is_complete()) {
            logger().trace("Running broad phase");
            {
                SCALABLE_CCD_GPU_PROFILE_POINT("Broad Phase");
                broad_phase.detect_overlaps_partial();
                gpuErrchk(hipDeviceSynchronize());
            }

            const Scalar earliest_toi_before = earliest_toi;
#ifdef SCALABLE_CCD_TOI_PER_QUERY
            std::vector<std::tuple<int, int, Scalar>> _collisions;
#endif

            logger().debug("Running narrow phase");
            {
                SCALABLE_CCD_GPU_PROFILE_POINT("Narrow Phase");

                narrow_phase<run_vf>(
                    d_vertices_t0, d_vertices_t1, d_edges, d_faces,
                    broad_phase.overlaps(), np_threads, max_iterations,
                    tolerance, min_distance, /*allow_zero_toi=*/true,
                    memory_handler,
#ifdef SCALABLE_CCD_TOI_PER_QUERY
                    _collisions,
#endif
                    earliest_toi);

                gpuErrchk(hipDeviceSynchronize());
            }

            if (earliest_toi < 1e-6) {
                logger().debug(
                    "Running narrow phase again (earliest_toi={:g})",
                    earliest_toi);
                earliest_toi = earliest_toi_before;

                SCALABLE_CCD_GPU_PROFILE_POINT("Narrow Phase");
                narrow_phase<run_vf>(
                    d_vertices_t0, d_vertices_t1, d_edges, d_faces,
                    broad_phase.overlaps(), np_threads, /*max_iterations=*/-1,
                    tolerance, /*ms=*/0.0, /*allow_zero_toi=*/false,
                    memory_handler,
#ifdef SCALABLE_CCD_TOI_PER_QUERY
                    _collisions,
#endif
                    earliest_toi);
                earliest_toi *= 0.8;

                gpuErrchk(hipDeviceSynchronize());
            }
        }
    }
} // namespace

Scalar ipc_ccd_strategy(
    const Eigen::MatrixXd& vertices_t0,
    const Eigen::MatrixXd& vertices_t1,
    const Eigen::MatrixXi& edges,
    const Eigen::MatrixXi& faces,
    const Scalar min_distance,
    const int max_iterations,
    const Scalar tolerance)
{
    assert(vertices_t0.rows() == vertices_t1.rows());
    assert(vertices_t0.cols() == vertices_t1.cols());
    assert(vertices_t0.cols() == 3);
    assert(edges.cols() == 2);
    assert(faces.cols() == 3);

    // --- Copy mesh to device ------------------------------------------------
    logger().trace("Copying mesh to device");

    const DeviceMatrix<Scalar> d_vertices_t0(vertices_t0);
    const DeviceMatrix<Scalar> d_vertices_t1(vertices_t1);
    const DeviceMatrix<int> d_edges(edges);
    const DeviceMatrix<int> d_faces(faces);

    // --- Construct boxes ----------------------------------------------------
    logger().trace("Constructing boxes");

    std::vector<AABB> vertex_boxes, edge_boxes, face_boxes;
    build_vertex_boxes(vertices_t0, vertices_t1, vertex_boxes, min_distance);
    build_edge_boxes(vertex_boxes, edges, edge_boxes);
    build_face_boxes(vertex_boxes, faces, face_boxes);

    const std::shared_ptr<DeviceAABBs> d_vertex_boxes =
        std::make_shared<DeviceAABBs>(vertex_boxes);
    const std::shared_ptr<DeviceAABBs> d_edge_boxes =
        std::make_shared<DeviceAABBs>(edge_boxes);
    const std::shared_ptr<DeviceAABBs> d_face_boxes =
        std::make_shared<DeviceAABBs>(face_boxes);

    // --- Run broad + narrow phase -------------------------------------------

    Scalar earliest_toi = 1.0;

    partial_ipc_ccd_strategy</*run_vf=*/true>(
        d_vertices_t0, d_vertices_t1, d_edges, d_faces, d_vertex_boxes,
        d_edge_boxes, d_face_boxes, min_distance, max_iterations, tolerance,
        earliest_toi);

    partial_ipc_ccd_strategy</*run_vf=*/false>(
        d_vertices_t0, d_vertices_t1, d_edges, d_faces, d_vertex_boxes,
        d_edge_boxes, d_face_boxes, min_distance, max_iterations, tolerance,
        earliest_toi);

    logger().debug("Earliest toi: {:g}", earliest_toi);

    return earliest_toi;
}

} // namespace scalable_ccd::cuda