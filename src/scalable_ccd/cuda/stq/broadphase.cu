#include "hip/hip_runtime.h"
#include "broadphase.cuh"

#include <scalable_ccd/config.hpp>
#include <scalable_ccd/cuda/stq/sweep.cuh>
#include <scalable_ccd/cuda/stq/util.cuh>
#include <scalable_ccd/cuda/utils/profiler.hpp>
#include <scalable_ccd/cuda/utils/device_variable.cuh>

#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#include <tbb/parallel_for.h>

namespace scalable_ccd::cuda::stq {

void BroadPhase::clear()
{
    *memory_handler = MemoryHandler();

    d_boxes.clear();
    d_boxes.shrink_to_fit();

    d_sm.clear();
    d_sm.shrink_to_fit();

    d_mini.clear();
    d_mini.shrink_to_fit();

    d_overlaps.clear();
    d_overlaps.shrink_to_fit();

    num_boxes_per_thread = 0;
    threads_per_block = 32;
    start_thread_id = 0;
    num_devices = 1;
}

const thrust::device_vector<cuda::stq::AABB>&
BroadPhase::build(const std::vector<cuda::stq::AABB>& boxes)
{
    logger().trace("Number of boxes: {:d}", boxes.size());

    if (memory_handler->MAX_OVERLAP_CUTOFF == 0) {
        memory_handler->MAX_OVERLAP_CUTOFF = boxes.size();
        logger().trace(
            "Setting MAX_OVERLAP_CUTOFF to {:d}",
            memory_handler->MAX_OVERLAP_CUTOFF);
    }

    if (memory_limit_GB) {
        logger().trace("Setting memory limit to {:d} GB", memory_limit_GB);
        memory_handler->limitGB = memory_limit_GB;
    }

    setup(device_init_id, smemSize, threads_per_block, num_boxes_per_thread);
    hipSetDevice(device_init_id);

    d_boxes = boxes; // copy to device
    d_sm.resize(boxes.size());
    d_mini.resize(boxes.size());

    // const Dimension axis = calc_sort_dimension();
    const Dimension axis = x;

    // Initialize d_sm and d_mini
    {
        SCALABLE_CCD_GPU_PROFILE_POINT("splitBoxes");
        splitBoxes<<<grid_dim_1d(), threads_per_block>>>(
            thrust::raw_pointer_cast(d_boxes.data()),
            thrust::raw_pointer_cast(d_sm.data()),
            thrust::raw_pointer_cast(d_mini.data()), d_boxes.size(), axis);
        gpuErrchk(hipDeviceSynchronize());
    }

    {
        SCALABLE_CCD_GPU_PROFILE_POINT("sortingBoxes");
        thrust::sort_by_key(
            thrust::device, d_sm.begin(), d_sm.end(), d_mini.begin(),
            sort_aabb_x());
        thrust::sort(
            thrust::device, d_boxes.begin(), d_boxes.end(), sort_aabb_x());
    }

    gpuErrchk(hipGetLastError());

    return d_boxes;
}

const thrust::device_vector<int2>& BroadPhase::detect_overlaps_partial()
{
    memory_handler->setOverlapSize();
    logger().trace(
        "Max overlap size: {:d} ({:g} GB)", memory_handler->MAX_OVERLAP_SIZE,
        memory_handler->MAX_OVERLAP_SIZE * sizeof(int2) / 1e9);
    logger().trace(
        "Max overlap cutoff: {:d}", memory_handler->MAX_OVERLAP_CUTOFF);

    // Device memory_handler to keep track of vars
    device_variable<MemoryHandler> d_memory_handler(*memory_handler);
    const int& real_count = memory_handler->real_count;

    do {
        // Allocate a large chunk of memory for overlaps
        // d_overlaps.resize(memory_handler->MAX_OVERLAP_CUTOFF);
        d_overlaps.resize(memory_handler->MAX_OVERLAP_SIZE);

        {
            SCALABLE_CCD_GPU_PROFILE_POINT("runSTQ");
            // This will be the actual number of overlaps
            device_variable<int> d_num_overlaps(0);
            device_variable<int> d_start(start_thread_id);
            // runSTQ<<<grid_dim_1d(), threads_per_block>>>(
            //     thrust::raw_pointer_cast(d_sm.data()),
            //     thrust::raw_pointer_cast(d_mini.data()),
            //     /*num_boxes=*/d_boxes.size(),
            //     thrust::raw_pointer_cast(d_overlaps.data()),
            //     thrust::raw_pointer_cast(d_num_overlaps.data()),
            //     thrust::raw_pointer_cast(d_start.data()),
            //     thrust::raw_pointer_cast(d_memory_handler.data()));

            runSAP<<<grid_dim_1d(), threads_per_block>>>(
                thrust::raw_pointer_cast(d_sm.data()),
                thrust::raw_pointer_cast(d_mini.data()),
                /*num_boxes=*/d_boxes.size(),
                thrust::raw_pointer_cast(d_overlaps.data()),
                d_num_overlaps.ptr(), d_start.ptr(), d_memory_handler.ptr());

            gpuErrchk(hipDeviceSynchronize());

            // Resize overlaps to actual size (keeps the capacity the same)
            d_overlaps.resize(d_num_overlaps);
        }

        *memory_handler = d_memory_handler;

        if (d_overlaps.size() < real_count) {
            logger().debug(
                "Found {:d} overlaps, but {:d} exist; re-running.",
                d_overlaps.size(), real_count);

            // Increase MAX_OVERLAP_SIZE (or decrease MAX_OVERLAP_CUTOFF)
            memory_handler->handleBroadPhaseOverflow(real_count);

            // Update memory handler on device
            d_memory_handler = *memory_handler;
        }
    } while (d_overlaps.size() < real_count);
    assert(real_count == d_overlaps.size());

    // Increase start_thread_id for next run
    start_thread_id += memory_handler->MAX_OVERLAP_CUTOFF;

    // Free up excess memory
    d_overlaps.shrink_to_fit();

    logger().debug(
        "Final count for device {:d}: {:d} ({:g} GB)", 0, d_overlaps.size(),
        d_overlaps.size() * sizeof(int2) / 1e9);
    logger().trace("Next threadstart {:d}", start_thread_id);

    return d_overlaps;
}

std::vector<std::pair<int, int>> BroadPhase::detect_overlaps()
{
    std::vector<std::pair<int, int>> overlaps;

    while (!is_complete()) {
        detect_overlaps_partial();

        const int n = overlaps.size();
        overlaps.resize(n + d_overlaps.size());

        gpuErrchk(hipMemcpy(
            &overlaps[n], thrust::raw_pointer_cast(d_overlaps.data()),
            d_overlaps.size() * sizeof(int2), hipMemcpyDeviceToHost));
    }

    logger().debug("Complete overlaps size {:d}", overlaps.size());

    return overlaps;
}

// ----------------------------------------------------------------------------

Dimension BroadPhase::calc_sort_dimension() const
{
    // mean of all box points (used to find best axis)
    thrust::device_vector<Scalar3> d_mean(1, make_Scalar3(0, 0, 0));
    calc_mean<<<grid_dim_1d(), threads_per_block, smemSize>>>(
        thrust::raw_pointer_cast(d_boxes.data()), d_boxes.size(),
        thrust::raw_pointer_cast(d_mean.data()));

    // temporary
    const Scalar3 mean = d_mean[0];
    logger().trace("mean: x {:.6f} y {:.6f} z {:.6f}", mean.x, mean.y, mean.z);

    // calculate variance and determine which axis to sort on
    device_variable<Scalar3> d_variance(make_Scalar3(0, 0, 0));

    calc_variance<<<grid_dim_1d(), threads_per_block, smemSize>>>(
        thrust::raw_pointer_cast(d_boxes.data()), d_boxes.size(),
        thrust::raw_pointer_cast(d_mean.data()), d_variance.ptr());
    hipDeviceSynchronize();

    const Scalar3 variance = d_variance;
    logger().trace(
        "var: x {:.6f} y {:.6f} z {:.6f}", variance.x, variance.y, variance.z);
    const Scalar max_variance =
        std::max({ variance.x, variance.y, variance.z });

    Dimension axis;
    if (max_variance == variance.x) {
        axis = x;
    } else if (max_variance == variance.y) {
        axis = y;
    } else {
        axis = z;
    }
    logger().trace("Axis: {:s}", axis == x ? "x" : (axis == y ? "y" : "z"));
    return axis;
}

} // namespace scalable_ccd::cuda::stq