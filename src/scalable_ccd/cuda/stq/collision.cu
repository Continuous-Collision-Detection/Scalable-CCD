#include "hip/hip_runtime.h"
#include <scalable_ccd/cuda/stq/collision.cuh>

namespace scalable_ccd::cuda::stq {

__device__ bool does_collide(const AABB& a, const AABB& b)
{
    return
        //    a.max.x >= b.min.x && a.min.x <= b.max.x &&
        a.max.y >= b.min.y && a.min.y <= b.max.y && a.max.z >= b.min.z
        && a.min.z <= b.max.z;
}

__device__ bool does_collide(const MiniBox& a, const MiniBox& b)
{
    return a.max.x >= b.min.x && a.min.x <= b.max.x && a.max.y >= b.min.y
        && a.min.y <= b.max.y;
}

__device__ bool covertex(const int3& a, const int3& b)
{

    return a.x == b.x || a.x == b.y || a.x == b.z || a.y == b.x || a.y == b.y
        || a.y == b.z || a.z == b.x || a.z == b.y || a.z == b.z;
}

__device__ void add_overlap(
    const int xid,
    const int yid,
    const int max_overlap_size,
    int2* overlaps,
    int* count)
{
    int i = atomicAdd(count, 1);

    if (i < max_overlap_size) {
        overlaps[i] = make_int2(xid, yid);
    }
}

__device__ void add_overlap(
    const int xid,
    const int yid,
    RawDeviceBuffer<int2>& overlaps,
    int& real_count)
{
    if (atomicAdd(&real_count, 1) < overlaps.capacity) {
        overlaps.push(make_int2(xid, yid));
    }
}

} // namespace scalable_ccd::cuda::stq