#include "hip/hip_runtime.h"
#include <scalable_ccd/config.hpp>
#include <scalable_ccd/cuda/stq/aabb.cuh>

#include <limits>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <tbb/parallel_for.h>

namespace scalable_ccd::cuda::stq {

__host__ __device__ bool is_face(const Aabb& x) { return x.vertexIds.z >= 0; };

__host__ __device__ bool is_face(const int3& vids) { return vids.z >= 0; };

__host__ __device__ bool is_edge(const Aabb& x)
{
    return x.vertexIds.z < 0 && x.vertexIds.y >= 0;
};

__host__ __device__ bool is_edge(const int3& vids)
{
    return vids.z < 0 && vids.y >= 0;
};

__host__ __device__ bool is_vertex(const Aabb& x)
{
    return x.vertexIds.z < 0 && x.vertexIds.y < 0;
};

__host__ __device__ bool is_vertex(const int3& vids)
{
    return vids.z < 0 && vids.y < 0;
};

__host__ __device__ bool is_valid_pair(const Aabb& a, const Aabb& b)
{
    return (is_vertex(a) && is_face(b)) || (is_face(a) && is_vertex(b))
        || (is_edge(a) && is_edge(b));
};

__host__ __device__ bool is_valid_pair(const int3& a, const int3& b)
{
    return (is_vertex(a) && is_face(b)) || (is_face(a) && is_vertex(b))
        || (is_edge(a) && is_edge(b));
};

void merge_local_boxes(
    const tbb::enumerable_thread_specific<std::vector<Aabb>>& storages,
    std::vector<Aabb>& boxes)
{
    size_t num_boxes = boxes.size();
    for (const auto& local_boxes : storages) {
        num_boxes += local_boxes.size();
    }
    // serial merge!
    boxes.reserve(num_boxes);
    for (const auto& local_boxes : storages) {
        boxes.insert(boxes.end(), local_boxes.begin(), local_boxes.end());
    }
}

float nextafter_up(float x)
{
    return nextafterf(x, x + std::numeric_limits<float>::max());
};
float nextafter_down(float x)
{
    return nextafterf(x, x - std::numeric_limits<float>::max());
};

void addEdges(
    const Eigen::MatrixXd& vertices_t0,
    const Eigen::MatrixXd& vertices_t1,
    const Eigen::MatrixXi& edges,
    Scalar inflation_radius,
    std::vector<Aabb>& boxes)
{
    tbb::enumerable_thread_specific<std::vector<Aabb>> storages;
    tbb::parallel_for(0, static_cast<int>(edges.rows()), 1, [&](int& i) {
        // for (unsigned long i = 0; i < edges.rows(); i++) {
        Eigen::MatrixXd edge_vertex0_t0 = vertices_t0.row(edges(i, 0));
        Eigen::MatrixXd edge_vertex1_t0 = vertices_t0.row(edges(i, 1));
        Eigen::MatrixXd edge_vertex0_t1 = vertices_t1.row(edges(i, 0));
        Eigen::MatrixXd edge_vertex1_t1 = vertices_t1.row(edges(i, 1));

        Eigen::MatrixXd points(4, edge_vertex0_t0.size());
        points.row(0) = edge_vertex0_t0;
        points.row(1) = edge_vertex1_t0;
        points.row(2) = edge_vertex0_t1;
        points.row(3) = edge_vertex1_t1;

        int vertexIds[3] = { edges(i, 0), edges(i, 1), -edges(i, 0) - 1 };
#ifdef SCALABLE_CCD_WITH_DOUBLE
        Eigen::Vector3d lower_bound =
            points.colwise().minCoeff().array() - inflation_radius;
        Eigen::Vector3d upper_bound =
            points.colwise().maxCoeff().array() + inflation_radius;
#else

    Eigen::MatrixXf lower_bound =
        points.colwise().minCoeff().unaryExpr(&nextafter_down).array() - nextafter_up(inflation_radius);
    Eigen::MatrixXf upper_bound =
        points.colwise().maxCoeff().unaryExpr(&nextafter_up).array() + nextafter_up(inflation_radius);
#endif
        auto& local_boxes = storages.local();
        local_boxes.emplace_back(
            boxes.size() + i, i, vertexIds, lower_bound.data(),
            upper_bound.data());
    });
    merge_local_boxes(storages, boxes);
}

void addVertices(
    const Eigen::MatrixXd& vertices_t0,
    const Eigen::MatrixXd& vertices_t1,
    Scalar inflation_radius,
    std::vector<Aabb>& boxes)
{
    tbb::enumerable_thread_specific<std::vector<Aabb>> storages;
    tbb::parallel_for(0, static_cast<int>(vertices_t0.rows()), 1, [&](int& i) {
        // for (unsigned long i = 0; i < vertices_t0.rows(); i++) {
        Eigen::MatrixXd vertex_t0 = vertices_t0.row(i);
        Eigen::MatrixXd vertex_t1 = vertices_t1.row(i);

        Eigen::MatrixXd points(2, vertex_t0.size());
        points.row(0) = vertex_t0;
        points.row(1) = vertex_t1;

        int vertexIds[3] = { i, -i - 1, -i - 1 };

#ifdef SCALABLE_CCD_WITH_DOUBLE
        Eigen::MatrixXd lower_bound =
            points.colwise().minCoeff().array() - inflation_radius;
        Eigen::MatrixXd upper_bound =
            points.colwise().maxCoeff().array() + inflation_radius;
#else

    Eigen::MatrixXf lower_bound =
        points.colwise().minCoeff().unaryExpr(&nextafter_down).array() - nextafter_up(inflation_radius);;
    Eigen::MatrixXf upper_bound =
    points.colwise().maxCoeff().unaryExpr(&nextafter_up).array() +  nextafter_up(inflation_radius);;
#endif
        auto& local_boxes = storages.local();
        local_boxes.emplace_back(
            boxes.size() + i, i, vertexIds, lower_bound.data(),
            upper_bound.data());
    });
    merge_local_boxes(storages, boxes);
}

void addFaces(
    const Eigen::MatrixXd& vertices_t0,
    const Eigen::MatrixXd& vertices_t1,
    const Eigen::MatrixXi& faces,
    Scalar inflation_radius,
    std::vector<Aabb>& boxes)
{
    tbb::enumerable_thread_specific<std::vector<Aabb>> storages;
    tbb::parallel_for(0, static_cast<int>(faces.rows()), 1, [&](int& i) {
        // for (unsigned long i = 0; i < faces.rows(); i++) {
        Eigen::MatrixXd face_vertex0_t0 = vertices_t0.row(faces(i, 0));
        Eigen::MatrixXd face_vertex1_t0 = vertices_t0.row(faces(i, 1));
        Eigen::MatrixXd face_vertex2_t0 = vertices_t0.row(faces(i, 2));
        Eigen::MatrixXd face_vertex0_t1 = vertices_t1.row(faces(i, 0));
        Eigen::MatrixXd face_vertex1_t1 = vertices_t1.row(faces(i, 1));
        Eigen::MatrixXd face_vertex2_t1 = vertices_t1.row(faces(i, 2));

        Eigen::MatrixXd points(6, face_vertex0_t0.size());
        points.row(0) = face_vertex0_t0;
        points.row(1) = face_vertex1_t0;
        points.row(2) = face_vertex2_t0;
        points.row(3) = face_vertex0_t1;
        points.row(4) = face_vertex1_t1;
        points.row(5) = face_vertex2_t1;

        int vertexIds[3] = { faces(i, 0), faces(i, 1), faces(i, 2) };

#ifdef SCALABLE_CCD_WITH_DOUBLE
        Eigen::Vector3d lower_bound = points.colwise().minCoeff().array()
            - static_cast<double>(inflation_radius);
        Eigen::Vector3d upper_bound = points.colwise().maxCoeff().array()
            + static_cast<double>(inflation_radius);
#else

    Eigen::MatrixXf lower_bound =
        points.colwise().minCoeff().unaryExpr(&nextafter_down).array() - nextafter_up(inflation_radius);;
    Eigen::MatrixXf upper_bound =
        points.colwise().maxCoeff().unaryExpr(&nextafter_up).array() + nextafter_up(inflation_radius);;
#endif
        auto& local_boxes = storages.local();
        local_boxes.emplace_back(
            boxes.size() + i, i, vertexIds, lower_bound.data(),
            upper_bound.data());
    });
    merge_local_boxes(storages, boxes);
};

} // namespace scalable_ccd::cuda::stq