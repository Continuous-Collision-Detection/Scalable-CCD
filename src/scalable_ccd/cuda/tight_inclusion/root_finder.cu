#include "hip/hip_runtime.h"
#include "root_finder.cuh"

#include <scalable_ccd/config.hpp>
#include <scalable_ccd/cuda/utils/limits.cuh>
#include <scalable_ccd/utils/logger.hpp>

#include <array>
#include <vector>

#include <cuda/semaphore>

using namespace std;

namespace scalable_ccd::cuda {

// this function do the bisection
__device__ IntervalPair::IntervalPair(const Interval& itv)
{
    Scalar c = (itv.first + itv.second) / 2;
    first.first = itv.first;
    first.second = c;
    second.first = c;
    second.second = itv.second;
}

__device__ bool sum_no_larger_1(const Scalar& num1, const Scalar& num2)
{
#ifdef SCALABLE_CCD_USE_DOUBLE
    if (num1 + num2 > 1 / (1 - DBL_EPSILON)) {
        return false;
    }
#else
    if (num1 + num2 > 1 / (1 - FLT_EPSILON)) {
        return false;
    }
#endif
    return true;
}

__device__ void compute_face_vertex_tolerance_memory_pool(
    CCDData& data_in, const CCDConfig& config)
{
    Scalar p000[3], p001[3], p011[3], p010[3], p100[3], p101[3], p111[3],
        p110[3];
    for (int i = 0; i < 3; i++) {
        p000[i] = data_in.v0s[i] - data_in.v1s[i];
        p001[i] = data_in.v0s[i] - data_in.v3s[i];
        p011[i] =
            data_in.v0s[i] - (data_in.v2s[i] + data_in.v3s[i] - data_in.v1s[i]);
        p010[i] = data_in.v0s[i] - data_in.v2s[i];
        p100[i] = data_in.v0e[i] - data_in.v1e[i];
        p101[i] = data_in.v0e[i] - data_in.v3e[i];
        p111[i] =
            data_in.v0e[i] - (data_in.v2e[i] + data_in.v3e[i] - data_in.v1e[i]);
        p110[i] = data_in.v0e[i] - data_in.v2e[i];
    }
    Scalar dl = 0;
    for (int i = 0; i < 3; i++) {
        dl = max(dl, fabs(p100[i] - p000[i]));
        dl = max(dl, fabs(p101[i] - p001[i]));
        dl = max(dl, fabs(p111[i] - p011[i]));
        dl = max(dl, fabs(p110[i] - p010[i]));
    }
    dl *= 3;
    data_in.tol[0] = config.co_domain_tolerance / dl;

    dl = 0;
    for (int i = 0; i < 3; i++) {
        dl = max(dl, fabs(p010[i] - p000[i]));
        dl = max(dl, fabs(p110[i] - p100[i]));
        dl = max(dl, fabs(p111[i] - p101[i]));
        dl = max(dl, fabs(p011[i] - p001[i]));
    }
    dl *= 3;
    data_in.tol[1] = config.co_domain_tolerance / dl;

    dl = 0;
    for (int i = 0; i < 3; i++) {
        dl = max(dl, fabs(p001[i] - p000[i]));
        dl = max(dl, fabs(p101[i] - p100[i]));
        dl = max(dl, fabs(p111[i] - p110[i]));
        dl = max(dl, fabs(p011[i] - p010[i]));
    }
    dl *= 3;
    data_in.tol[2] = config.co_domain_tolerance / dl;
}
__device__ void compute_edge_edge_tolerance_memory_pool(
    CCDData& data_in, const CCDConfig& config)
{
    Scalar p000[3], p001[3], p011[3], p010[3], p100[3], p101[3], p111[3],
        p110[3];
    for (int i = 0; i < 3; i++) {
        p000[i] = data_in.v0s[i] - data_in.v2s[i];
        p001[i] = data_in.v0s[i] - data_in.v3s[i];
        p011[i] = data_in.v1s[i] - data_in.v3s[i];
        p010[i] = data_in.v1s[i] - data_in.v2s[i];
        p100[i] = data_in.v0e[i] - data_in.v2e[i];
        p101[i] = data_in.v0e[i] - data_in.v3e[i];
        p111[i] = data_in.v1e[i] - data_in.v3e[i];
        p110[i] = data_in.v1e[i] - data_in.v2e[i];
    }
    Scalar dl = 0;
    for (int i = 0; i < 3; i++) {
        dl = max(dl, fabs(p100[i] - p000[i]));
        dl = max(dl, fabs(p101[i] - p001[i]));
        dl = max(dl, fabs(p111[i] - p011[i]));
        dl = max(dl, fabs(p110[i] - p010[i]));
    }
    dl *= 3;
    data_in.tol[0] = config.co_domain_tolerance / dl;

    dl = 0;
    for (int i = 0; i < 3; i++) {
        dl = max(dl, fabs(p010[i] - p000[i]));
        dl = max(dl, fabs(p110[i] - p100[i]));
        dl = max(dl, fabs(p111[i] - p101[i]));
        dl = max(dl, fabs(p011[i] - p001[i]));
    }
    dl *= 3;
    data_in.tol[1] = config.co_domain_tolerance / dl;

    dl = 0;
    for (int i = 0; i < 3; i++) {
        dl = max(dl, fabs(p001[i] - p000[i]));
        dl = max(dl, fabs(p101[i] - p100[i]));
        dl = max(dl, fabs(p111[i] - p110[i]));
        dl = max(dl, fabs(p011[i] - p010[i]));
    }
    dl *= 3;
    data_in.tol[2] = config.co_domain_tolerance / dl;
}

std::array<Scalar, 3> get_numerical_error(
    const std::vector<std::array<Scalar, 3>>& vertices,
    const bool& check_vf,
    const bool use_ms)
{
    Scalar eefilter;
    Scalar vffilter;
    if (!use_ms) {
#ifdef SCALABLE_CCD_USE_DOUBLE
        eefilter = 6.217248937900877e-15;
        vffilter = 6.661338147750939e-15;
#else
        eefilter = 3.337861e-06;
        vffilter = 3.576279e-06;
#endif
    } else // using minimum separation
    {
#ifdef SCALABLE_CCD_USE_DOUBLE
        eefilter = 7.105427357601002e-15;
        vffilter = 7.549516567451064e-15;
#else
        eefilter = 3.814698e-06;
        vffilter = 4.053116e-06;
#endif
    }

    Scalar xmax = fabs(vertices[0][0]);
    Scalar ymax = fabs(vertices[0][1]);
    Scalar zmax = fabs(vertices[0][2]);
    for (int i = 0; i < vertices.size(); i++) {
        if (xmax < fabs(vertices[i][0])) {
            xmax = fabs(vertices[i][0]);
        }
        if (ymax < fabs(vertices[i][1])) {
            ymax = fabs(vertices[i][1]);
        }
        if (zmax < fabs(vertices[i][2])) {
            zmax = fabs(vertices[i][2]);
        }
    }
    Scalar delta_x = xmax > 1 ? xmax : 1;
    Scalar delta_y = ymax > 1 ? ymax : 1;
    Scalar delta_z = zmax > 1 ? zmax : 1;
    std::array<Scalar, 3> result;
    if (!check_vf) {
        result[0] = delta_x * delta_x * delta_x * eefilter;
        result[1] = delta_y * delta_y * delta_y * eefilter;
        result[2] = delta_z * delta_z * delta_z * eefilter;
    } else {
        result[0] = delta_x * delta_x * delta_x * vffilter;
        result[1] = delta_y * delta_y * delta_y * vffilter;
        result[2] = delta_z * delta_z * delta_z * vffilter;
    }
    return result;
}

__device__ __host__ void
get_numerical_error_vf_memory_pool(CCDData& data_in, bool use_ms)
{
    Scalar vffilter;
    //   bool use_ms = false;
    if (!use_ms) {
#ifdef SCALABLE_CCD_USE_DOUBLE
        vffilter = 6.661338147750939e-15;
#else
        vffilter = 3.576279e-06;
#endif
    } else {
#ifdef SCALABLE_CCD_USE_DOUBLE
        vffilter = 7.549516567451064e-15;
#else
        vffilter = 4.053116e-06;
#endif
    }
    Scalar xmax = fabs(data_in.v0s[0]);
    Scalar ymax = fabs(data_in.v0s[1]);
    Scalar zmax = fabs(data_in.v0s[2]);

    xmax = max(xmax, fabs(data_in.v1s[0]));
    ymax = max(ymax, fabs(data_in.v1s[1]));
    zmax = max(zmax, fabs(data_in.v1s[2]));

    xmax = max(xmax, fabs(data_in.v2s[0]));
    ymax = max(ymax, fabs(data_in.v2s[1]));
    zmax = max(zmax, fabs(data_in.v2s[2]));

    xmax = max(xmax, fabs(data_in.v3s[0]));
    ymax = max(ymax, fabs(data_in.v3s[1]));
    zmax = max(zmax, fabs(data_in.v3s[2]));

    xmax = max(xmax, fabs(data_in.v0e[0]));
    ymax = max(ymax, fabs(data_in.v0e[1]));
    zmax = max(zmax, fabs(data_in.v0e[2]));

    xmax = max(xmax, fabs(data_in.v1e[0]));
    ymax = max(ymax, fabs(data_in.v1e[1]));
    zmax = max(zmax, fabs(data_in.v1e[2]));

    xmax = max(xmax, fabs(data_in.v2e[0]));
    ymax = max(ymax, fabs(data_in.v2e[1]));
    zmax = max(zmax, fabs(data_in.v2e[2]));

    xmax = max(xmax, fabs(data_in.v3e[0]));
    ymax = max(ymax, fabs(data_in.v3e[1]));
    zmax = max(zmax, fabs(data_in.v3e[2]));

    xmax = max(xmax, Scalar(1));
    ymax = max(ymax, Scalar(1));
    zmax = max(zmax, Scalar(1));

    data_in.err[0] = xmax * xmax * xmax * vffilter;
    data_in.err[1] = ymax * ymax * ymax * vffilter;
    data_in.err[2] = zmax * zmax * zmax * vffilter;
    return;
}

__device__ __host__ void
get_numerical_error_ee_memory_pool(CCDData& data_in, bool use_ms)
{
    Scalar vffilter;
    //   bool use_ms = false;
    if (!use_ms) {

#ifdef SCALABLE_CCD_USE_DOUBLE
        vffilter = 6.217248937900877e-15;
#else
        vffilter = 3.337861e-06;
#endif
    } else {
#ifdef SCALABLE_CCD_USE_DOUBLE
        vffilter = 7.105427357601002e-15;
#else
        vffilter = 3.814698e-06;
#endif
    }
    Scalar xmax = fabs(data_in.v0s[0]);
    Scalar ymax = fabs(data_in.v0s[1]);
    Scalar zmax = fabs(data_in.v0s[2]);

    xmax = max(xmax, fabs(data_in.v1s[0]));
    ymax = max(ymax, fabs(data_in.v1s[1]));
    zmax = max(zmax, fabs(data_in.v1s[2]));

    xmax = max(xmax, fabs(data_in.v2s[0]));
    ymax = max(ymax, fabs(data_in.v2s[1]));
    zmax = max(zmax, fabs(data_in.v2s[2]));

    xmax = max(xmax, fabs(data_in.v3s[0]));
    ymax = max(ymax, fabs(data_in.v3s[1]));
    zmax = max(zmax, fabs(data_in.v3s[2]));

    xmax = max(xmax, fabs(data_in.v0e[0]));
    ymax = max(ymax, fabs(data_in.v0e[1]));
    zmax = max(zmax, fabs(data_in.v0e[2]));

    xmax = max(xmax, fabs(data_in.v1e[0]));
    ymax = max(ymax, fabs(data_in.v1e[1]));
    zmax = max(zmax, fabs(data_in.v1e[2]));

    xmax = max(xmax, fabs(data_in.v2e[0]));
    ymax = max(ymax, fabs(data_in.v2e[1]));
    zmax = max(zmax, fabs(data_in.v2e[2]));

    xmax = max(xmax, fabs(data_in.v3e[0]));
    ymax = max(ymax, fabs(data_in.v3e[1]));
    zmax = max(zmax, fabs(data_in.v3e[2]));

    xmax = max(xmax, Scalar(1));
    ymax = max(ymax, Scalar(1));
    zmax = max(zmax, Scalar(1));

    data_in.err[0] = xmax * xmax * xmax * vffilter;
    data_in.err[1] = ymax * ymax * ymax * vffilter;
    data_in.err[2] = zmax * zmax * zmax * vffilter;
    return;
}

__device__ void BoxPrimatives::calculate_tuv(const MP_unit& unit)
{
    if (b[0] == 0) { // t0
        t = unit.itv[0].first;
    } else { // t1
        t = unit.itv[0].second;
    }

    if (b[1] == 0) { // u0
        u = unit.itv[1].first;
    } else { // u1
        u = unit.itv[1].second;
    }

    if (b[2] == 0) { // v0
        v = unit.itv[2].first;
    } else { // v1
        v = unit.itv[2].second;
    }
}

__device__ Scalar calculate_vf(const CCDData& data_in, const BoxPrimatives& bp)
{
    Scalar v, pt, t0, t1, t2;
    v = (data_in.v0e[bp.dim] - data_in.v0s[bp.dim]) * bp.t
        + data_in.v0s[bp.dim];
    t0 = (data_in.v1e[bp.dim] - data_in.v1s[bp.dim]) * bp.t
        + data_in.v1s[bp.dim];
    t1 = (data_in.v2e[bp.dim] - data_in.v2s[bp.dim]) * bp.t
        + data_in.v2s[bp.dim];
    t2 = (data_in.v3e[bp.dim] - data_in.v3s[bp.dim]) * bp.t
        + data_in.v3s[bp.dim];
    pt = (t1 - t0) * bp.u + (t2 - t0) * bp.v + t0;
    return (v - pt);
}

__device__ Scalar calculate_ee(const CCDData& data_in, const BoxPrimatives& bp)
{
    Scalar edge0_vertex0 = (data_in.v0e[bp.dim] - data_in.v0s[bp.dim]) * bp.t
        + data_in.v0s[bp.dim];
    Scalar edge0_vertex1 = (data_in.v1e[bp.dim] - data_in.v1s[bp.dim]) * bp.t
        + data_in.v1s[bp.dim];
    Scalar edge1_vertex0 = (data_in.v2e[bp.dim] - data_in.v2s[bp.dim]) * bp.t
        + data_in.v2s[bp.dim];
    Scalar edge1_vertex1 = (data_in.v3e[bp.dim] - data_in.v3s[bp.dim]) * bp.t
        + data_in.v3s[bp.dim];
    Scalar result = ((edge0_vertex1 - edge0_vertex0) * bp.u + edge0_vertex0)
        - ((edge1_vertex1 - edge1_vertex0) * bp.v + edge1_vertex0);

    return result;
}

inline __device__ bool Origin_in_vf_inclusion_function_memory_pool(
    const CCDData& data_in, MP_unit& unit, Scalar& true_tol, bool& box_in)
{
    box_in = true;
    true_tol = 0.0;
    BoxPrimatives bp;
    Scalar vmin = ::cuda::numeric_limits<Scalar>::max();
    Scalar vmax = -::cuda::numeric_limits<Scalar>::max();
    Scalar value;
    for (bp.dim = 0; bp.dim < 3; bp.dim++) {
        vmin = ::cuda::numeric_limits<Scalar>::max();
        vmax = -::cuda::numeric_limits<Scalar>::max();
        for (int i = 0; i < 2; i++) {
            for (int j = 0; j < 2; j++) {
                for (int k = 0; k < 2; k++) {
                    bp.b[0] = i;
                    bp.b[1] = j;
                    bp.b[2] = k; // 100
                    bp.calculate_tuv(unit);
                    value = calculate_vf(data_in, bp);
                    vmin = min(vmin, value);
                    vmax = max(vmax, value);
                }
            }
        }

        // get the min and max in one dimension
        true_tol = max(true_tol, vmax - vmin);

        if (vmin - data_in.ms > data_in.err[bp.dim]
            || vmax + data_in.ms < -data_in.err[bp.dim]) {
            return false;
        }

        if (vmin + data_in.ms < -data_in.err[bp.dim]
            || vmax - data_in.ms > data_in.err[bp.dim]) {
            box_in = false;
        }
    }
    return true;
}

inline __device__ bool Origin_in_ee_inclusion_function_memory_pool(
    const CCDData& data_in, MP_unit& unit, Scalar& true_tol, bool& box_in)
{
    box_in = true;
    true_tol = 0.0;
    BoxPrimatives bp;
    Scalar vmin = ::cuda::numeric_limits<Scalar>::max();
    Scalar vmax = -::cuda::numeric_limits<Scalar>::max();
    Scalar value;
    for (bp.dim = 0; bp.dim < 3; bp.dim++) {
        vmin = ::cuda::numeric_limits<Scalar>::max();
        vmax = -::cuda::numeric_limits<Scalar>::max();
        for (int i = 0; i < 2; i++) {
            for (int j = 0; j < 2; j++) {
                for (int k = 0; k < 2; k++) {
                    bp.b[0] = i;
                    bp.b[1] = j;
                    bp.b[2] = k; // 100
                    bp.calculate_tuv(unit);
                    value = calculate_ee(data_in, bp);
                    vmin = min(vmin, value);
                    vmax = max(vmax, value);
                }
            }
        }

        // get the min and max in one dimension
        true_tol = max(true_tol, vmax - vmin);

        if (vmin - data_in.ms > data_in.err[bp.dim]
            || vmax + data_in.ms < -data_in.err[bp.dim]) {
            return false;
        }

        if (vmin + data_in.ms < -data_in.err[bp.dim]
            || vmax - data_in.ms > data_in.err[bp.dim]) {
            box_in = false;
        }
    }
    return true;
}

// === the memory pool method =================================================

__global__ void compute_vf_tolerance_memory_pool(
    CCDData* data, CCDConfig* config, const int query_size)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx >= query_size)
        return;

    // release the mutex here before real calculations
    config[0].mutex.release();

    compute_face_vertex_tolerance_memory_pool(data[tx], config[0]);

    data[tx].nbr_checks = 0;
    get_numerical_error_vf_memory_pool(data[tx], config[0].use_ms);
}

__global__ void compute_ee_tolerance_memory_pool(
    CCDData* data, CCDConfig* config, const int query_size)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx >= query_size)
        return;

    // release the mutex here before real calculations
    config[0].mutex.release();

    compute_edge_edge_tolerance_memory_pool(data[tx], config[0]);

    data[tx].nbr_checks = 0;
    get_numerical_error_ee_memory_pool(data[tx], config[0].use_ms);
}

__global__ void initialize_memory_pool(MP_unit* units, int query_size)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx >= query_size)
        return;
    units[tx].init(tx);
}
__device__ int split_dimension_memory_pool(const CCDData& data, Scalar width[3])
{ // clarified in queue.h
    int split = 0;
    Scalar res[3];
    res[0] = width[0] / data.tol[0];
    res[1] = width[1] / data.tol[1];
    res[2] = width[2] / data.tol[2];
    if (res[0] >= res[1] && res[0] >= res[2]) {
        split = 0;
    }
    if (res[1] >= res[0] && res[1] >= res[2]) {
        split = 1;
    }
    if (res[2] >= res[1] && res[2] >= res[0]) {
        split = 2;
    }
    return split;
}

inline __device__ bool bisect_vf_memory_pool(
    const MP_unit& unit,
    int split,
    CCDConfig* config,
#ifdef SCALABLE_CCD_TOI_PER_QUERY
    Scalar data_toi,
#endif
    MP_unit* out)
{
    IntervalPair halves(unit.itv[split]); // bisected

    if (halves.first.first >= halves.first.second) {
        // valid_nbr = 0;
        return true;
    }
    if (halves.second.first >= halves.second.second) {
        // valid_nbr = 0;
        return true;
    }
    // bisected[0] = unit;
    // bisected[1] = unit;
    // valid_nbr = 1;

    int unit_id = atomicInc(&config[0].mp_end, config[0].unit_size - 1);
    out[unit_id] = unit;
    out[unit_id].itv[split] = halves.first;

    if (split == 0) {
#ifndef SCALABLE_CCD_TOI_PER_QUERY
        if (halves.second.first <= config[0].toi) {
#else
        if (halves.second.first <= data_toi) {
#endif
            unit_id = atomicInc(&config[0].mp_end, config[0].unit_size - 1);
            out[unit_id] = unit;
            out[unit_id].itv[split] = halves.second;
        }
    } else if (split == 1) {
        if (sum_no_larger_1(
                halves.second.first,
                unit.itv[2].first)) // check if u+v<=1
        {
            unit_id = atomicInc(&config[0].mp_end, config[0].unit_size - 1);
            out[unit_id] = unit;
            out[unit_id].itv[1] = halves.second;
            // valid_nbr = 2;
        }
    } else if (split == 2) {
        if (sum_no_larger_1(
                halves.second.first,
                unit.itv[1].first)) // check if u+v<=1
        {
            unit_id = atomicInc(&config[0].mp_end, config[0].unit_size - 1);
            out[unit_id] = unit;
            out[unit_id].itv[2] = halves.second;
            // valid_nbr = 2;
        }
    }
    return false;
}
inline __device__ bool bisect_ee_memory_pool(
    const MP_unit& unit,
    int split,
    CCDConfig* config,
#ifdef SCALABLE_CCD_TOI_PER_QUERY
    Scalar data_toi,
#endif
    MP_unit* out)
{
    IntervalPair halves(unit.itv[split]); // bisected

    if (halves.first.first >= halves.first.second) {
        // valid_nbr = 0;
        return true;
    }
    if (halves.second.first >= halves.second.second) {
        // valid_nbr = 0;
        return true;
    }
    // bisected[0] = unit;
    // bisected[1] = unit;
    // valid_nbr = 1;

    int unit_id = atomicInc(&config[0].mp_end, config[0].unit_size - 1);
    out[unit_id] = unit;
    out[unit_id].itv[split] = halves.first;

    if (split == 0) // split the time interval
    {
#ifndef SCALABLE_CCD_TOI_PER_QUERY
        if (halves.second.first <= config[0].toi) {
#else
        if (halves.second.first <= data_toi) {
#endif
            unit_id = atomicInc(&config[0].mp_end, config[0].unit_size - 1);
            out[unit_id] = unit;
            out[unit_id].itv[split] = halves.second;
        }
    } else {

        unit_id = atomicInc(&config[0].mp_end, config[0].unit_size - 1);
        out[unit_id] = unit;
        out[unit_id].itv[split] = halves.second;
        // valid_nbr = 2;
    }

    return false;
}

inline __device__ void mutex_update_min(
    ::cuda::binary_semaphore<::cuda::thread_scope_device>& mutex,
    Scalar& value,
    const Scalar& compare)
{
    mutex.acquire();
    value =
        compare < value ? compare : value; // if compare is smaller, update it
    mutex.release();
}

__global__ void vf_ccd_memory_pool(
    MP_unit* units, int query_size, CCDData* data, CCDConfig* config)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx >= config[0].mp_remaining)
        return;

    //   bool allow_zero_toi = true;
    int qid = (tx + config[0].mp_start) % config[0].unit_size;

    Scalar widths[3];
    bool condition;
    // int split;

    MP_unit units_in = units[qid];
    int box_id = units_in.query_id;
    CCDData data_in = data[box_id];

    atomicAdd(&data[box_id].nbr_checks, 1);

    const Scalar time_left = units_in.itv[0].first; // the time of this unit

// if the time is larger than toi, return
#ifndef SCALABLE_CCD_TOI_PER_QUERY
    if (time_left >= config[0].toi) {
        return;
    }
#else
    if (time_left >= data_in.toi)
        return;
#endif
    // if (results[box_id] > 0)
    // { // if it is sure that have root, then no need to check
    // 	return;
    // }
    if (config[0].max_iter >= 0
        && data_in.nbr_checks > config[0].max_iter) // max checks
    {
        // if (!config[0].overflow_flag)
        //   atomicAdd(&config[0].overflow_flag, 1);
        return;
    } else if (config[0].mp_remaining > config[0].unit_size / 2) // overflow
    {
        if (!config[0].overflow_flag)
            atomicAdd(&config[0].overflow_flag, 1);
        return;
    }

    Scalar true_tol = 0;
    bool box_in;

    const bool zero_in = Origin_in_vf_inclusion_function_memory_pool(
        data_in, units_in, true_tol, box_in);
    if (zero_in) {
        widths[0] = units_in.itv[0].second - units_in.itv[0].first;
        widths[1] = units_in.itv[1].second - units_in.itv[1].first;
        widths[2] = units_in.itv[2].second - units_in.itv[2].first;

        // Condition 1
        condition = widths[0] <= data_in.tol[0] && widths[1] <= data_in.tol[1]
            && widths[2] <= data_in.tol[2];
        if (condition) {
            mutex_update_min(config[0].mutex, config[0].toi, time_left);
            // results[box_id] = 1;

#ifdef SCALABLE_CCD_TOI_PER_QUERY
            mutex_update_min(config[0].mutex, data[box_id].toi, time_left);
#endif
            return;
        }
        // Condition 2, the box is inside the epsilon box, have a root, return
        // true; condition = units_in.box_in;

        if (box_in && (config[0].allow_zero_toi || time_left > 0)) {
            mutex_update_min(config[0].mutex, config[0].toi, time_left);
            // results[box_id] = 1;

#ifdef SCALABLE_CCD_TOI_PER_QUERY
            mutex_update_min(config[0].mutex, data[box_id].toi, time_left);
#endif
            return;
        }

        // Condition 3, real tolerance is smaller than the input tolerance,
        // return true
        condition = true_tol <= config->co_domain_tolerance;
        if (condition && (config[0].allow_zero_toi || time_left > 0)) {
            mutex_update_min(config[0].mutex, config[0].toi, time_left);
            // results[box_id] = 1;

#ifdef SCALABLE_CCD_TOI_PER_QUERY
            mutex_update_min(config[0].mutex, data[box_id].toi, time_left);
#endif
            return;
        }
        const int split = split_dimension_memory_pool(data_in, widths);

#ifndef SCALABLE_CCD_TOI_PER_QUERY
        const bool sure_in =
            bisect_vf_memory_pool(units_in, split, config, units);
#else
        const bool sure_in =
            bisect_vf_memory_pool(units_in, split, config, data_in.toi, units);
#endif

        if (sure_in) // in this case, the interval is too small that overflow
                     // happens. it should be rare to happen
        {
            mutex_update_min(config[0].mutex, config[0].toi, time_left);
            // results[box_id] = 1;

#ifdef SCALABLE_CCD_TOI_PER_QUERY
            mutex_update_min(config[0].mutex, data[box_id].toi, time_left);
#endif
            return;
        }
    }
}
__global__ void ee_ccd_memory_pool(
    MP_unit* units, int query_size, CCDData* data, CCDConfig* config)
{
    //   bool allow_zero_toi = true;

    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx >= config[0].mp_remaining)
        return;

    int qid = (tx + config[0].mp_start) % config[0].unit_size;

    Scalar widths[3];
    bool condition;

    MP_unit units_in = units[qid];
    int box_id = units_in.query_id;
    CCDData data_in = data[box_id];

    atomicAdd(&data[box_id].nbr_checks, 1);

    const Scalar time_left = units_in.itv[0].first; // the time of this unit

// if the time is larger than toi, return
#ifndef SCALABLE_CCD_TOI_PER_QUERY
    if (time_left >= config[0].toi) {
        return;
    }
#else
    if (time_left >= data_in.toi)
        return;
#endif
    // if (results[box_id] > 0)
    // { // if it is sure that have root, then no need to check
    // 	return;
    // }
    if (config[0].max_iter >= 0
        && data_in.nbr_checks > config[0].max_iter) // max checks
    {
        // if (!config[0].overflow_flag)
        //   atomicAdd(&config[0].overflow_flag, 1);
        return;
    } else if (config[0].mp_remaining > config[0].unit_size / 2) // overflow
    {
        if (!config[0].overflow_flag)
            atomicAdd(&config[0].overflow_flag, 1);
        return;
    }

    Scalar true_tol = 0;
    bool box_in;

    const bool zero_in = Origin_in_ee_inclusion_function_memory_pool(
        data_in, units_in, true_tol, box_in);
    if (zero_in) {
        widths[0] = units_in.itv[0].second - units_in.itv[0].first;
        widths[1] = units_in.itv[1].second - units_in.itv[1].first;
        widths[2] = units_in.itv[2].second - units_in.itv[2].first;

        // Condition 1
        condition = widths[0] <= data_in.tol[0] && widths[1] <= data_in.tol[1]
            && widths[2] <= data_in.tol[2];
        if (condition) {
            mutex_update_min(config[0].mutex, config[0].toi, time_left);
            // results[box_id] = 1;

#ifdef SCALABLE_CCD_TOI_PER_QUERY
            mutex_update_min(config[0].mutex, data[box_id].toi, time_left);
#endif
            return;
        }
        // Condition 2, the box is inside the epsilon box, have a root, return
        // true;
        if (box_in && (config[0].allow_zero_toi || time_left > 0)) {
            mutex_update_min(config[0].mutex, config[0].toi, time_left);
            // results[box_id] = 1;

#ifdef SCALABLE_CCD_TOI_PER_QUERY
            mutex_update_min(config[0].mutex, data[box_id].toi, time_left);
#endif
            return;
        }

        // Condition 3, real tolerance is smaller than the input tolerance,
        // return true
        condition = true_tol <= config->co_domain_tolerance;
        if (condition && (config[0].allow_zero_toi || time_left > 0)) {
            mutex_update_min(config[0].mutex, config[0].toi, time_left);
            // results[box_id] = 1;

#ifdef SCALABLE_CCD_TOI_PER_QUERY
            mutex_update_min(config[0].mutex, data[box_id].toi, time_left);
#endif
            return;
        }
        const int split = split_dimension_memory_pool(data_in, widths);

#ifndef SCALABLE_CCD_TOI_PER_QUERY
        const bool sure_in =
            bisect_ee_memory_pool(units_in, split, config, units);
#else
        const bool sure_in =
            bisect_ee_memory_pool(units_in, split, config, data_in.toi, units);
#endif

        if (sure_in) // in this case, the interval is too small that overflow
                     // happens. it should be rare to happen
        {
            mutex_update_min(config[0].mutex, config[0].toi, time_left);
            // results[box_id] = 1;

#ifdef SCALABLE_CCD_TOI_PER_QUERY
            mutex_update_min(config[0].mutex, data[box_id].toi, time_left);
#endif

            return;
        }
    }
}

__global__ void shift_queue_pointers(CCDConfig* config)
{
    config[0].mp_start += config[0].mp_remaining;
    config[0].mp_start = config[0].mp_start % config[0].unit_size;
    config[0].mp_remaining = (config[0].mp_end - config[0].mp_start);
    config[0].mp_remaining = config[0].mp_remaining < 0
        ? config[0].mp_end + config[0].unit_size - config[0].mp_start
        : config[0].mp_remaining;
}

bool run_memory_pool_ccd(
    thrust::device_vector<CCDData>& d_data_list,
    std::shared_ptr<MemoryHandler> memory_handler,
    const bool is_edge,
    std::vector<int>& result_list,
    const int parallel_nbr,
    const int max_iter,
    const Scalar tol,
    const bool use_ms,
    const bool allow_zero_toi,
    Scalar& toi)
{
    const int nbr = d_data_list.size();

    // memory_handler->setUnitSize(/*constraint=*/sizeof(CCDConfig));

    // int *res = new int[nbr];
    CCDConfig* config = new CCDConfig[1];
    // config[0].err_in[0] =
    //     -1; // the input error bound calculate from the AABB of the whole
    //     mesh
    config[0].co_domain_tolerance = tol; // tolerance of the co-domain
    // config[0].max_t = 1;                  // the upper bound of the time

    // interval
    config[0].toi = toi;
    config[0].mp_end = nbr;
    config[0].mp_start = 0;
    config[0].mp_remaining = nbr;
    config[0].overflow_flag = 0;
    config[0].unit_size =
        memory_handler
            ->MAX_UNIT_SIZE; // std::min(1024 * nbr, int(5e7)); // 2.0 * nbr;
    config[0].use_ms = use_ms;
    config[0].allow_zero_toi = allow_zero_toi;
    config[0].max_iter = max_iter;

    // int *d_res;
    MP_unit* d_units;
    CCDConfig* d_config;

    size_t unit_size = sizeof(MP_unit) * config[0].unit_size;
    logger().debug("unit_size: {:d}", config[0].unit_size);
    logger().debug("unit_size (bytes): {:d}", unit_size);
    logger().debug(
        "allocatable (bytes): {:d}", memory_handler->__getAllocatable());
    // size_t result_size = sizeof(int) * nbr;

    // hipMalloc(&d_res, result_size);
    gpuErrchk(hipMalloc(&d_units, unit_size));
    gpuErrchk(hipMalloc(&d_config, sizeof(CCDConfig)));
    gpuErrchk(hipMemcpy(
        d_config, config, sizeof(CCDConfig), hipMemcpyHostToDevice));
    gpuErrchk(hipGetLastError());
    // Timer timer;
    // timer.start();
    logger().trace("nbr: {:d}, parallel_nbr: {:d}", nbr, parallel_nbr);
    initialize_memory_pool<<<nbr / parallel_nbr + 1, parallel_nbr>>>(
        d_units, nbr);
    gpuErrchk(hipDeviceSynchronize());

    if (is_edge) {
        compute_ee_tolerance_memory_pool<<<
            nbr / parallel_nbr + 1, parallel_nbr>>>(
            thrust::raw_pointer_cast(d_data_list.data()), d_config, nbr);
    } else {
        compute_vf_tolerance_memory_pool<<<
            nbr / parallel_nbr + 1, parallel_nbr>>>(
            thrust::raw_pointer_cast(d_data_list.data()), d_config, nbr);
    }
    gpuErrchk(hipDeviceSynchronize());

    logger().trace("MAX_QUERIES: {:d}", memory_handler->MAX_QUERIES);
    logger().trace("sizeof(Scalar) {:d}", sizeof(Scalar));

    int nbr_per_loop = nbr;
    // int start = 0;
    // int end = 0;

    logger().trace("Queue size t0: {:d}", nbr_per_loop);
    while (nbr_per_loop > 0) {
        if (is_edge) {
            ee_ccd_memory_pool<<<
                nbr_per_loop / parallel_nbr + 1, parallel_nbr>>>(
                d_units, nbr, thrust::raw_pointer_cast(d_data_list.data()),
                d_config);
        } else {
            vf_ccd_memory_pool<<<
                nbr_per_loop / parallel_nbr + 1, parallel_nbr>>>(
                d_units, nbr, thrust::raw_pointer_cast(d_data_list.data()),
                d_config);
        }
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipGetLastError());
        shift_queue_pointers<<<1, 1>>>(d_config);
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipMemcpy(
            &nbr_per_loop, &d_config[0].mp_remaining, sizeof(int),
            hipMemcpyDeviceToHost));
        // hipMemcpy(&start, &d_config[0].mp_start, sizeof(int),
        //            hipMemcpyDeviceToHost);
        // hipMemcpy(&end, &d_config[0].mp_end, sizeof(int),
        // hipMemcpyDeviceToHost); hipMemcpy(&toi, &d_config[0].toi,
        // sizeof(Scalar),
        //            hipMemcpyDeviceToHost);
        // logger().trace("toi {}", toi);
        // logger().trace("toi {:.4f}",  toi);
        // logger().trace("Start {:d}, End {:d}, Queue size: {:d}",  start, end,
        // nbr_per_loop);
        gpuErrchk(hipGetLastError());
        logger().trace("Queue size: {:d}", nbr_per_loop);
    }
    hipDeviceSynchronize();
    // double tt = timer.getElapsedTimeInMicroSec();
    // run_time += tt / 1000.0f;
    gpuErrchk(hipGetLastError());

    // hipMemcpy(res, d_res, result_size, hipMemcpyDeviceToHost);
    gpuErrchk(hipMemcpy(
        &toi, &d_config[0].toi, sizeof(Scalar), hipMemcpyDeviceToHost));
    int overflow;
    gpuErrchk(hipMemcpy(
        &overflow, &d_config[0].overflow_flag, sizeof(int),
        hipMemcpyDeviceToHost));
    if (overflow) {
        return true;
    }

    gpuErrchk(hipFree(d_units));
    gpuErrchk(hipFree(d_config));

    // for (size_t i = 0; i < nbr; i++) {
    //   result_list[i] = res[i];
    // }

    // delete[] res;
    delete[] config;
    hipError_t ct = hipGetLastError();
    logger().trace(
        "\n******************\n{}\n******************", hipGetErrorString(ct));

#ifdef SCALABLE_CCD_TOI_PER_QUERY
    CCDData* data_list = new CCDData[d_data_list.size()];
    // CCDConfig *config = new CCDConfig[1];
    gpuErrchk(hipMemcpy(
        data_list, d_data_list, sizeof(CCDData) * d_data_list.size(),
        hipMemcpyDeviceToHost));
    // std::vector<std::pair<std::string, std::string>> symbolic_tois;
    int tpq_cnt = 0;
    for (size_t i = 0; i < d_data_list.size(); i++) {
        cuda::stq::Rational ra(data_list[i].toi);
        if (data_list[i].toi > 1)
            continue;
        tpq_cnt++;
        // symbolic_tois.emplace_back(ra.get_numerator_str(),
        //                            ra.get_denominator_str());
        // auto pair = make_pair(ra.get_numerator_str(),
        // ra.get_denominator_str());
        std::string triple[4] = { std::to_string(data_list[i].aid),
                                  std::to_string(data_list[i].bid),
                                  ra.get_numerator_str(),
                                  ra.get_denominator_str() };
        // if (data_list[i].toi <= .00000382)
        //   printf("not one toi %s, %s, %e\n", triple[0].c_str(),
        //   triple[1].c_str(),
        //          data_list[i].toi);
        r.j_object["toi_per_query"].push_back(triple);
    }
    logger().trace("tpq_cnt: {:d}", tpq_cnt);
    free(data_list);
    gpuErrchk(hipDeviceSynchronize());
    // json jtmp(symbolic_tois.begin(), symbolic_tois.end());
    // std::cout << jtmp.dump(4) << std::endl;
    // r.j_object.insert(jtmp.begin(), jtmp.end());
    // r.j_object.push_back(r.j_object.end(), jtmp.begin(), jtmp.end());
    // r.j_object.push_back(symbolic_tois);
    //  symbolic_tois.end());

    // json j_vec(falseNegativePairs);
    // r.j_object.insert(r.j_object.end(), symbolic_tois.begin(),
    //                   symbolic_tois.end());

    // std::ofstream o(outputFilePath);
    // o << std::setw(4) << j_vec << std::endl;
    // auto outputFilename = std::filesystem::path(std::to_string(iter) +
    // ".json"); outputFilename = outputFolder / outputFilename; std::ofstream
    // o(outputFilename); o << std::setw(4) << j << std::endl;
#endif

    return false;
}

} // namespace scalable_ccd::cuda
