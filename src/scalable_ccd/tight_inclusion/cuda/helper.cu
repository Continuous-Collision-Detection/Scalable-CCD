#include "hip/hip_runtime.h"
#include <scalable_ccd/config.hpp>

#include <ccdgpu/CType.cuh>
#include <ccdgpu/helper.cuh>

#include <fstream>
#include <iostream>

// #include <gputi/book.h>
// #include <gputi/io.h>
#include <ccdgpu/root_finder.cuh>
#include <ccdgpu/timer.hpp>
#include <scalable_ccd/stq/cuda/io.cuh>

#include <ccdgpu/record.hpp>
#include <scalable_ccd/stq/cuda/memory.cuh>
#include <scalable_ccd/stq/cuda/broadphase.cuh>

#include <spdlog/spdlog.h>

using namespace std;
using namespace stq::gpu;

namespace ccd::gpu {

#define gpuErrchk(ans)                                                         \
    {                                                                          \
        gpuAssert((ans), __FILE__, __LINE__);                                  \
    }

inline void
gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess) {
        spdlog::error(
            "GPUassert: {} {} {:d}", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

// Allocates and copies data to GPU
template <typename T> T* copy_to_gpu(const T* cpu_data, const int size)
{
    T* gpu_data;
    gpuErrchk(hipMalloc((void**)&gpu_data, sizeof(T) * size));
    gpuErrchk(hipMemcpy(
        gpu_data, cpu_data, sizeof(T) * size, hipMemcpyHostToDevice));
    return gpu_data;
}

__global__ void split_overlaps(
    const int2* const overlaps,
    const stq::gpu::Aabb* const boxes,
    int N,
    int2* vf_overlaps,
    int2* ee_overlaps,
    int* vf_count,
    int* ee_count)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= N)
        return;

    int minner = min(overlaps[tid].x, overlaps[tid].y);
    int maxxer = max(overlaps[tid].x, overlaps[tid].y);
    int3 avids = boxes[minner].vertexIds;
    int3 bvids = boxes[maxxer].vertexIds;

    if (is_vertex(avids) && is_face(bvids)) {
        int i = atomicAdd(vf_count, 1);
        vf_overlaps[i].x = minner;
        vf_overlaps[i].y = maxxer;
    } else if (is_edge(avids) && is_edge(bvids)) {
        int j = atomicAdd(ee_count, 1);
        ee_overlaps[j].x = minner;
        ee_overlaps[j].y = maxxer;
    } else
        assert(false);
}

__global__ void addData(
    const int2* const overlaps,
    const stq::gpu::Aabb* const boxes,
    const Scalar* const V0,
    const Scalar* const V1,
    int Vrows,
    int N,
    Scalar ms,
    CCDData* data)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid >= N)
        return;

    data[tid].ms = ms;

    // spdlog::trace("vf_count {:d}, ee_count {:d}", *vf_count, *ee_count);

    int minner = min(overlaps[tid].x, overlaps[tid].y);
    int maxxer = max(overlaps[tid].x, overlaps[tid].y);
    int3 avids = boxes[minner].vertexIds;
    int3 bvids = boxes[maxxer].vertexIds;

#ifdef SCALABLE_CCD_TOI_PER_QUERY
    data[tid].toi = std::numeric_limits<Scalar>::infinity();
    // data[tid].id = shift + tid;
    data[tid].aid = minner;
    data[tid].bid = maxxer;
#endif

    if (is_vertex(avids) && is_face(bvids)) {
        for (size_t i = 0; i < 3; i++) {
            data[tid].v0s[i] = V0[avids.x + i * Vrows];
            data[tid].v1s[i] = V0[bvids.x + i * Vrows];
            data[tid].v2s[i] = V0[bvids.y + i * Vrows];
            data[tid].v3s[i] = V0[bvids.z + i * Vrows];
            data[tid].v0e[i] = V1[avids.x + i * Vrows];
            data[tid].v1e[i] = V1[bvids.x + i * Vrows];
            data[tid].v2e[i] = V1[bvids.y + i * Vrows];
            data[tid].v3e[i] = V1[bvids.z + i * Vrows];
        }
    } else if (is_edge(avids) && is_edge(bvids)) {
        for (size_t i = 0; i < 3; i++) {
            data[tid].v0s[i] = V0[avids.x + i * Vrows];
            data[tid].v1s[i] = V0[avids.y + i * Vrows];
            data[tid].v2s[i] = V0[bvids.x + i * Vrows];
            data[tid].v3s[i] = V0[bvids.y + i * Vrows];
            data[tid].v0e[i] = V1[avids.x + i * Vrows];
            data[tid].v1e[i] = V1[avids.y + i * Vrows];
            data[tid].v2e[i] = V1[bvids.x + i * Vrows];
            data[tid].v3e[i] = V1[bvids.y + i * Vrows];
        }
    } else
        assert(false);
}

void run_narrowphase(
    int2* d_overlaps,
    Aabb* d_boxes,
    stq::gpu::MemHandler* memhandle,
    int count,
    Scalar* d_vertices_t0,
    Scalar* d_vertices_t1,
    int Vrows,
    int threads,
    int max_iter,
    Scalar tol,
    Scalar ms,
    bool allow_zero_toi,
    vector<int>& result_list,
    Scalar& toi,
    Record& r)
{
    bool use_ms = ms > 0;

    int* d_vf_count;
    int* d_ee_count;

    int2* d_vf_overlaps;
    int2* d_ee_overlaps;

    int start_id = 0;
    int size = count;
    memhandle->MAX_QUERIES = size;

    // double tavg = 0;
    // double tmp_tall = 0;

    size_t remain;
    spdlog::trace("remain {:d}, size {:d}", remain, size);
    while ((remain = size - start_id) > 0
#ifndef SCALABLE_CCD_TOI_PER_QUERY
           && toi > 0
#endif
    ) {
        spdlog::trace("remain {:d}, start_id {:d}", remain, start_id);

        int overflow = 1; // run at least once
        int itr = 0;
        int tmp_nbr;
        while (overflow) {
            tmp_nbr = std::min(remain, memhandle->MAX_QUERIES);

            spdlog::debug("itr {:d}", itr);
            if (itr == 0) {
                memhandle->handleNarrowPhase(tmp_nbr);
            } else {
                memhandle->handleOverflow(tmp_nbr);
            }

            itr++;

            r.Start("splitOverlaps", /*gpu=*/true);
            gpuErrchk(hipMalloc((void**)&d_vf_count, sizeof(int)));
            gpuErrchk(hipMalloc((void**)&d_ee_count, sizeof(int)));

            gpuErrchk(hipMemset(d_vf_count, 0, sizeof(int)));
            gpuErrchk(hipMemset(d_ee_count, 0, sizeof(int)));

            gpuErrchk(
                hipMalloc((void**)&d_vf_overlaps, sizeof(int2) * tmp_nbr));
            gpuErrchk(
                hipMalloc((void**)&d_ee_overlaps, sizeof(int2) * tmp_nbr));

            split_overlaps<<<tmp_nbr / threads + 1, threads>>>(
                d_overlaps + start_id, d_boxes, tmp_nbr, d_vf_overlaps,
                d_ee_overlaps, d_vf_count, d_ee_count);
            gpuErrchk(hipDeviceSynchronize());
            r.Stop();

            r.Start("createDataList", /*gpu=*/true);
            int vf_size;
            int ee_size;
            gpuErrchk(hipMemcpy(
                &vf_size, d_vf_count, sizeof(int), hipMemcpyDeviceToHost));
            gpuErrchk(hipMemcpy(
                &ee_size, d_ee_count, sizeof(int), hipMemcpyDeviceToHost));
            spdlog::trace("vf_size {} ee_size {}", vf_size, ee_size);

            CCDData* d_ee_data_list;
            CCDData* d_vf_data_list;

            size_t ee_data_size = sizeof(CCDData) * ee_size;
            size_t vf_data_size = sizeof(CCDData) * vf_size;

            gpuErrchk(hipMalloc((void**)&d_ee_data_list, ee_data_size));
            gpuErrchk(hipMalloc((void**)&d_vf_data_list, vf_data_size));
            spdlog::trace("ee_data_size {:d}", ee_data_size);
            spdlog::trace("vf_data_size {:d}", vf_data_size);

            addData<<<vf_size / threads + 1, threads>>>(
                d_vf_overlaps, d_boxes, d_vertices_t0, d_vertices_t1, Vrows,
                vf_size, ms, d_vf_data_list);
            gpuErrchk(hipDeviceSynchronize());
            addData<<<ee_size / threads + 1, threads>>>(
                d_ee_overlaps, d_boxes, d_vertices_t0, d_vertices_t1, Vrows,
                ee_size, ms, d_ee_data_list);
            gpuErrchk(hipDeviceSynchronize());

            r.Stop();

            gpuErrchk(hipFree(d_vf_overlaps));
            gpuErrchk(hipFree(d_ee_overlaps));
            gpuErrchk(hipFree(d_vf_count));
            gpuErrchk(hipFree(d_ee_count));

            spdlog::trace("vf_size {:d}, ee_size {:d}", vf_size, ee_size);

            // int size = count;
            // spdlog::trace("data loaded, size {}", queries.size());
            spdlog::trace("data loaded, size {}", size);

            // result_list.resize(size);

            int parallel = 64;
            spdlog::trace("run_memory_pool_ccd using {:d} threads", parallel);
            r.Start("run_memory_pool_ccd (narrowphase)", /*gpu=*/true);
            // toi = 1;
            run_memory_pool_ccd(
                d_vf_data_list, memhandle, vf_size,
                /*is_edge_edge=*/false, result_list, parallel, max_iter, tol,
                use_ms, allow_zero_toi, toi, overflow, r);

            gpuErrchk(hipDeviceSynchronize());

            r.Stop();
            if (overflow) // rerun
            {
                spdlog::warn("overflow after vf");
                gpuErrchk(hipFree(d_ee_data_list));
                continue;
            }
            spdlog::trace("toi after vf {:e}", toi);
            // spdlog::trace("time after vf {:.6f}",  tmp_tall);
            r.Start("run_memory_pool_ccd (narrowphase)", /*gpu=*/true);
            run_memory_pool_ccd(
                d_ee_data_list, memhandle, ee_size,
                /*is_edge_edge=*/true, result_list, parallel, max_iter, tol,
                use_ms, allow_zero_toi, toi, overflow, r);
            gpuErrchk(hipDeviceSynchronize());
            r.Stop();
            spdlog::trace("toi after ee {:e}", toi);
            if (overflow)
                spdlog::warn("overflow after ee");
        }

        start_id += tmp_nbr;
    }
}

void run_ccd(
    const vector<Aabb> boxes,
    stq::gpu::MemHandler* memhandle,
    const Eigen::MatrixXd& vertices_t0,
    const Eigen::MatrixXd& vertices_t1,
    Record& r,
    int N,
    int& nbox,
    int& parallel,
    int& devcount,
    int& limitGB,
    vector<pair<int, int>>& overlaps,
    vector<int>& result_list,
    bool& allow_zero_toi,
    Scalar& ms,
    Scalar& toi)
{
    toi = 1;
    bool use_ms = ms > 0;

    int tidstart = 0;

    int bpthreads = 32; // HARDCODING THREADS FOR NOW
    int npthreads = 1024;

    int2* d_overlaps;
    int* d_count;
    size_t tot_count = 0;
    while (N > tidstart && toi > 0) {

        spdlog::trace("Next loop: N {:d}, tidstart {:d}", N, tidstart);

        r.Start("runBroadPhase", /*gpu=*/true);
        runBroadPhase(
            boxes.data(), memhandle, N, nbox, overlaps, d_overlaps, d_count,
            bpthreads, tidstart, devcount, limitGB);
        r.Stop();

        spdlog::trace("First run end {:d}", tidstart);
        // memhandle->increaseOverlapCutoff(2);
        spdlog::trace("Next cutoff {:d}", memhandle->MAX_OVERLAP_CUTOFF);

        spdlog::trace("Threads now {:d}", npthreads);

        r.Start("copyBoxesToGpu", /*gpu=*/true);

        int count;
        gpuErrchk(
            hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));
        tot_count += count;
        spdlog::trace("Count {:d}", count);

        Aabb* d_boxes = copy_to_gpu(boxes.data(), boxes.size());
        r.Stop();

        r.Start("copyVerticesToGpu", /*gpu=*/true);
        spdlog::trace("Copying vertices");
        double* d_vertices_t0 =
            copy_to_gpu(vertices_t0.data(), vertices_t0.size());
        double* d_vertices_t1 =
            copy_to_gpu(vertices_t1.data(), vertices_t1.size());
        r.Stop();
        int Vrows = vertices_t0.rows();
        assert(Vrows == vertices_t1.rows());

        int max_iter = -1;
        Scalar tolerance = 1e-6;

        run_narrowphase(
            d_overlaps, d_boxes, memhandle, count, d_vertices_t0, d_vertices_t1,
            Vrows, npthreads, max_iter, tolerance, ms, allow_zero_toi,
            result_list, toi, r);
        gpuErrchk(hipGetLastError());

        gpuErrchk(hipFree(d_count));
        gpuErrchk(hipFree(d_overlaps));
        gpuErrchk(hipFree(d_boxes));
        gpuErrchk(hipFree(d_vertices_t0));
        gpuErrchk(hipFree(d_vertices_t1));

        gpuErrchk(hipGetLastError());

        hipDeviceSynchronize();
    }
    spdlog::info("Total count {:d}", tot_count);
    spdlog::info("LimitGB {:d}", memhandle->limitGB);
}

void construct_static_collision_candidates(
    const Eigen::MatrixXd& V,
    const Eigen::MatrixXi& E,
    const Eigen::MatrixXi& F,
    vector<pair<int, int>>& overlaps,
    vector<stq::gpu::Aabb>& boxes,
    double inflation_radius)
{
    construct_continuous_collision_candidates(
        V, V, E, F, overlaps, boxes, inflation_radius);
}

void construct_continuous_collision_candidates(
    const Eigen::MatrixXd& V0,
    const Eigen::MatrixXd& V1,
    const Eigen::MatrixXi& E,
    const Eigen::MatrixXi& F,
    vector<pair<int, int>>& overlaps,
    vector<stq::gpu::Aabb>& boxes,
    double inflation_radius)
{
    constructBoxes(V0, V1, E, F, boxes, -1, inflation_radius);
    int N = boxes.size();
    int nbox = 0;
    int devcount = 1;

    int2* d_overlaps;
    int* d_count;
    int bpthreads = 32;
    int start_id = 0;
    int limitGB = 0;
    stq::gpu::MemHandler* memhandle = new stq::gpu::MemHandler();
    while (N > start_id) {
        runBroadPhase(
            boxes.data(), memhandle, N, nbox, overlaps, d_overlaps, d_count,
            bpthreads, start_id, devcount, limitGB);
        gpuErrchk(hipDeviceSynchronize());
    }

    spdlog::trace("Overlaps size {:d}", overlaps.size());
    hipFree(d_overlaps);
    hipFree(d_count);
}

Scalar compute_toi_strategy(
    const Eigen::MatrixXd& V0,
    const Eigen::MatrixXd& V1,
    const Eigen::MatrixXi& E,
    const Eigen::MatrixXi& F,
    int max_iter,
    Scalar min_distance,
    Scalar tolerance)
{
    vector<stq::gpu::Aabb> boxes;
    constructBoxes(V0, V1, E, F, boxes);
    spdlog::trace("Finished constructing");
    int N = boxes.size();
    int nbox = 0;
    int devcount = 1;

    stq::gpu::MemHandler* memhandle = new stq::gpu::MemHandler();

    vector<pair<int, int>> overlaps;
    vector<int> result_list;

    // BROADPHASE
    int2* d_overlaps;
    int* d_count;
    int bpthreads = 32;
    int npthreads = 1024;
    int start_id = 0;
    int limitGB = 0;

    json j;
    Record r(j);

    Scalar earliest_toi = 1.0;

    while (N > start_id) {

        runBroadPhase(
            boxes.data(), memhandle, N, nbox, overlaps, d_overlaps, d_count,
            bpthreads, start_id, devcount, limitGB);

        // copy overlap count
        int count;
        gpuErrchk(
            hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));
        spdlog::trace("Count {:d}", count);

        // Allocate boxes to GPU
        Aabb* d_boxes = copy_to_gpu(boxes.data(), boxes.size());

        spdlog::trace("Copying vertices");
        double* d_vertices_t0 = copy_to_gpu(V0.data(), V0.size());
        double* d_vertices_t1 = copy_to_gpu(V1.data(), V1.size());

        int Vrows = V0.rows();
        assert(Vrows == V1.rows());

        run_narrowphase(
            d_overlaps, d_boxes, memhandle, count, d_vertices_t0, d_vertices_t1,
            Vrows, npthreads, /*max_iter=*/max_iter,
            /*tol=*/tolerance,
            /*ms=*/min_distance, /*allow_zero_toi=*/true, result_list,
            earliest_toi, r);

        if (earliest_toi < 1e-6) {
            run_narrowphase(
                d_overlaps, d_boxes, memhandle, count, d_vertices_t0,
                d_vertices_t1, Vrows, npthreads, /*max_iter=*/-1,
                /*tol=*/tolerance,
                /*ms=*/0.0, /*allow_zero_toi=*/false, result_list, earliest_toi,
                r);
            earliest_toi *= 0.8;
        }

        gpuErrchk(hipDeviceSynchronize());

        gpuErrchk(hipFree(d_count));
        gpuErrchk(hipFree(d_overlaps));
        gpuErrchk(hipFree(d_boxes));
        gpuErrchk(hipFree(d_vertices_t0));
        gpuErrchk(hipFree(d_vertices_t1));

        gpuErrchk(hipDeviceSynchronize());
    }

    return earliest_toi;
}

} // namespace ccd::gpu