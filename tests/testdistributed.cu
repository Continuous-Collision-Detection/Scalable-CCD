#include "hip/hip_runtime.h"
#include <stq/gpu/broad_phase.cuh>
#include <stq/gpu/groundtruth.h>
#include <stq/gpu/utils.cuh>

#include <vector>
#include <iostream>
#include <vector>
#include <numeric>
#include <string>
#include <functional>
#include <cmath>

#include <thrust/sort.h>
#include <thrust/execution_policy.h>

#include <tbb/parallel_for.h>
#include <tbb/blocked_range.h>
#include <tbb/enumerable_thread_specific.h>
#include <tbb/global_control.h>
#include <tbb/concurrent_vector.h>

using namespace std;

__global__ void
square_sum(int* d_in, int* d_out, int* d_count, int N, int start, int end)
{
    int tid = start + threadIdx.x + blockIdx.x * blockDim.x;
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= end || tid >= N)
        return;

    if (d_in[tid] % 5 == 0) {
        // d_out[gid] = d_in[tid] * d_in[tid];
        int i = atomicAdd(d_count, 1);
        d_out[i] = d_in[tid] * d_in[tid];
    } else {
        // int i = atomicAdd(d_count, 1);
        // d_out[i] = 5;
    }
}

void merge_local(
    const tbb::enumerable_thread_specific<vector<int>>& storages,
    std::vector<int>& overlaps)
{
    overlaps.clear();
    size_t num_overlaps = overlaps.size();
    for (const auto& local_overlaps : storages) {
        num_overlaps += local_overlaps.size();
    }
    // serial merge!
    overlaps.reserve(num_overlaps);
    for (const auto& local_overlaps : storages) {
        overlaps.insert(
            overlaps.end(), local_overlaps.begin(), local_overlaps.end());
    }
}

void run_sweep_multigpu(int N, int devcount)
{
    vector<int> squareSums;

    int in[N];
    for (int i = 0; i < N; i++)
        in[i] = N - i;

    cout << "default threads " << tbb::info::default_concurrency() << endl;
    // tbb::global_control
    // thread_limiter(tbb::global_control::max_allowed_parallelism, 2);
    tbb::enumerable_thread_specific<vector<int>> storages;

    int device_init_id = 0;

    // int smemSize;
    // setup(device_init_id, smemSize, threads, nbox);

    hipSetDevice(device_init_id);

    int* d_in;

    hipMalloc((void**)&d_in, sizeof(int) * N);

    hipMemcpy(d_in, in, sizeof(int) * N, hipMemcpyHostToDevice);

    int threads = 1024;
    dim3 block(threads);
    int grid_dim_1d = (N / threads + 1);
    dim3 grid(grid_dim_1d);

    try {
        thrust::sort(thrust::device, d_in, d_in + N);
    } catch (thrust::system_error& e) {
        printf("Error: %s \n", e.what());
    }
    hipDeviceSynchronize();

    int devices_count;
    hipGetDeviceCount(&devices_count);
    // devices_count-=2;
    devices_count = devcount ? devcount : devices_count;
    int range = ceil((float)N / devices_count);
    printf("range: %i\n", range);

    tbb::parallel_for(0, devices_count, 1, [&](int& device_id) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, device_id);
        printf(
            "%s -> unifiedAddressing = %d\n", prop.name,
            prop.unifiedAddressing);

        hipSetDevice(device_id);

        int is_able;

        for (int i = 0; i < devices_count; i++) {
            hipDeviceCanAccessPeer(&is_able, device_id, i);
            if (is_able) {
                hipDeviceEnablePeerAccess(i, 0);
            } else if (i != device_id)
                printf("Device %i cant access Device %i\n", device_id, i);
        }

        gpuErrchk(hipGetLastError());

        int range_start = range * device_id;
        int range_end = range * (device_id + 1);
        printf("device_id: %i [%i, %i)\n", device_id, range_start, range_end);

        int* d_in_solo;
        hipMalloc((void**)&d_in_solo, sizeof(int) * N);
        // if (device_id == device_init_id )
        hipMemcpy(d_in_solo, d_in, sizeof(int) * N, hipMemcpyDefault);

        // // turn off peer access for write variables
        sleep(1);
        for (int i = 0; i < devices_count; i++) {
            hipDeviceCanAccessPeer(&is_able, device_id, i);
            if (is_able) {
                hipDeviceDisablePeerAccess(i);
            } else if (i != device_id)
                printf("Device %i cant access Device %i\n", device_id, i);
        }
        sleep(1);

        int* d_out;
        hipMalloc((void**)&d_out, sizeof(int) * range);
        hipMemset(d_out, 0, sizeof(int) * range);

        int* d_count;
        hipMalloc((void**)&d_count, sizeof(int) * 1);
        hipMemset(d_count, 0, sizeof(int) * 1);

        square_sum<<<grid, block>>>(
            d_in_solo, d_out, d_count, N, range_start, range_end);
        gpuErrchk(hipDeviceSynchronize());

        int count;
        gpuErrchk(
            hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));
        printf("count for device %i : %i\n", device_id, count);
        hipFree(d_out);
        hipMalloc((void**)&d_out, sizeof(int) * count);
        hipMemset(d_out, -1, sizeof(int) * count);

        hipMemset(d_count, 0, sizeof(int) * 1);

        square_sum<<<grid, block>>>(
            d_in_solo, d_out, d_count, N, range_start, range_end);
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(
            hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));
        printf("count2 for device %i : %i\n", device_id, count);

        int* out = (int*)malloc(sizeof(int) * count);
        gpuErrchk(hipMemcpy(
            out, d_out, sizeof(int) * count, hipMemcpyDeviceToHost));

        auto& local_overlaps = storages.local();

        for (size_t i = 0; i < count; i++) {
            local_overlaps.emplace_back(out[i]);
        }

        printf(
            "Total(filt.) overlaps for devid %i: %i\n", device_id,
            local_overlaps.size());
        // delete [] overlaps;
        // free(overlaps);

        // // free(counter);
        // // free(counter);
        // hipFree(d_overlaps);
        // hipFree(d_count);
        // // hipFree(d_b);
        // // hipFree(d_r);
        // hipDeviceReset();
    }); // end tbb for loop

    merge_local(storages, squareSums);

    int sum = accumulate(squareSums.begin(), squareSums.end(), 0);
    printf("\nFinal result: %i\n", sum);
    printf("Final result size: %i\n", squareSums.size());
    printf("\n");
    for (int i = 0; i < squareSums.size(); i++) {
        printf("%i ", squareSums[i]);
    }
    printf("\n");
}

int main(int argc, char** argv)
{
    int N = 1;
    int devcount = 0;

    int o;
    while ((o = getopt(argc, argv, "n:d:")) != -1) {
        switch (o) {
        case 'n':
            N = atoi(optarg);
            break;
        case 'd':
            devcount = atoi(optarg);
            break;
        }
    }

    run_sweep_multigpu(N, devcount);
}
